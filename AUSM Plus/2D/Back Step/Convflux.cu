#include "hip/hip_runtime.h"
#include "ausmPlus.h"
#include <math.h>

__global__ void convectiveflux(cell *domain, double *R, double *gammma)
{
	int y=threadIdx.x;
	int x=blockIdx.x;
	int ourFlag=(int)domain[x].flag;
	if(ourFlag==0 || ourFlag==4)
	{
		//Calculating the critical speed of sound for all the four sides/faces and the element itself
		double a_s[2];

		//Element
		a_s[0]=sqrt(2*(gammma[0]-1)/(gammma[0]+1)*(domain[x].stateVar[3]+(gammma[0]-1)*(domain[x].stateVar[3]\
			-0.5*(pow(domain[x].stateVar[1],2)+pow(domain[x].stateVar[2],2))/domain[x].stateVar[0]))/domain[x].stateVar[0]);
		//Side/face
		a_s[1]=sqrt(2*(gammma[0]-1)/(gammma[0]+1)*(domain[x].temp_var[y][3]+(gammma[0]-1)*(domain[x].temp_var[y][3]\
			-0.5*(pow(domain[x].temp_var[y][1],2)+pow(domain[x].temp_var[y][2],2))/domain[x].temp_var[y][0]))/domain[x].temp_var[y][0]);

		//speed for the boundary calculation
		a_s[0]=pow(a_s[0],2)/max(a_s[0],abs(sqrt(pow(domain[x].stateVar[1]/domain[x].stateVar[0],2)+pow(domain[x].stateVar[2]/domain[x].stateVar[0],2))));
		a_s[1]=pow(a_s[1],2)/max(a_s[1],abs(sqrt(pow(domain[x].temp_var[y][1]/domain[x].temp_var[y][0],2)+pow(domain[x].temp_var[y][2]/domain[x].temp_var[y][0],2))));

		//Speed of sound at facial interface
		double a_mid=min(a_s[0],a_s[1]);

		//Pressure
		double pressplus=(*gammma-1)*(domain[x].stateVar[3]-0.5*(pow(domain[x].stateVar[1],2)+pow(domain[x].stateVar[2],2))/domain[x].stateVar[0]);
		double pressminus=(*gammma-1)*(domain[x].temp_var[y][3]-0.5*(pow(domain[x].temp_var[y][1],2)+pow(domain[x].temp_var[y][2],2))/domain[x].temp_var[y][0]);

		//Machnumber of contravarient velocity(V=u*nx+v*ny)
		double mach_one=(domain[x].stateVar[1]/domain[x].stateVar[0]*domain[x].norms[y][0]+domain[x].stateVar[2]/domain[x].stateVar[0]*domain[x].norms[y][1])/a_mid;
		double mach_two=(domain[x].temp_var[y][1]/domain[x].temp_var[y][0]*domain[x].norms[y][0]+domain[x].temp_var[y][2]/domain[x].temp_var[y][0]*domain[x].norms[y][1])/a_mid;

		double split_mach_one,split_mach_two;
		if(abs(mach_one)>=1)
			split_mach_one=0.5*(mach_one+abs(mach_one));
		else
			split_mach_one=(0.25*pow(mach_one+1,2))+((1.0/8.0)*(pow(pow(mach_one,2.0)-1.0,2.0)));

		if(abs(mach_two)>=1)
			split_mach_two=0.5*(mach_two-abs(mach_two));
		else
			split_mach_two=-0.25*pow(mach_two-1.0,2.0)-1.0/8.0*pow(pow(mach_two,2.0)-1.0,2.0);
	
			double split_mach=split_mach_one+split_mach_two;
		
		for (int i = 0; i < 4; ++i)
		{
			domain[x].convflux[y][i]=a_mid*(0.5*(split_mach+abs(split_mach))*domain[x].stateVar[i]+0.5*(split_mach-abs(split_mach))\
				*domain[x].temp_var[y][i])*sqrt(pow(domain[x].nodes[y][0]-domain[x].nodes[(y+1)%4][0],2)+pow(domain[x].nodes[y][1]-domain[x].nodes[(y+1)%4][1],2));
		}
		domain[x].convflux[y][3]+=a_mid*(0.5*(split_mach+abs(split_mach))*pressplus+0.5*(split_mach-abs(split_mach))*pressminus)\
		*sqrt(pow(domain[x].nodes[y][0]-domain[x].nodes[(y+1)%4][0],2)+pow(domain[x].nodes[y][1]-domain[x].nodes[(y+1)%4][1],2));
	}
}
