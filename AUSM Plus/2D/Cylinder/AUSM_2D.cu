#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <omp.h>
#include "ausmPlus.h"

using namespace std;

cell::cell(float *initial)
{
	//omp_set_nested(1);
	//omp_set_num_threads(4);
	//#pragma omp parallel for
	stateVar[0]=initial[0];
	stateVar[1]=0.0;
	stateVar[2]=0.0;
	stateVar[3]=(gammma-1)*(initial[3]-0.5*(powf(stateVar[1],2)+powf(stateVar[2],2))/stateVar[0]);
}

cell::cell(){}

__global__ void evaluate(cell *domain,float deltat)
{
	int x=blockIdx.x;
	int y=threadIdx.x;
	float vol=0.5*abs((domain[x].nodes[0][0]-domain[x].nodes[1][0])*(domain[x].nodes[0][1]+domain[x].nodes[1][1])+\
		(domain[x].nodes[1][0]-domain[x].nodes[2][0])*(domain[x].nodes[1][1]+domain[x].nodes[2][1])+\
		(domain[x].nodes[2][0]-domain[x].nodes[0][0])*(domain[x].nodes[2][1]+domain[x].nodes[0][1]));
	if(domain[x].flag==0 || domain[x].flag==4 || domain[x].flag==2)
	{	
		
		domain[x].stateVar[y]-=deltat*(domain[x].convflux[0][y]+domain[x].convflux[1][y]+domain[x].convflux[2][y]\
		-(domain[x].diffflux[0][y]+domain[x].diffflux[1][y]+domain[x].diffflux[2][y]))/vol;

		if(y==1)
			domain[x].stateVar[y]-=deltat*(domain[x].presflux[0][0]+domain[x].presflux[1][0]+domain[x].presflux[2][0])/vol;
		if(y==2)
		domain[x].stateVar[y]-=deltat*(domain[x].presflux[0][1]+domain[x].presflux[1][1]+domain[x].presflux[2][1])/vol;
	}
}

__global__ void Boundary(cell *domain,float *initial)
{
	int x=blockIdx.x;
	int y=threadIdx.x;
	int note=-8;
	int next=-8;

	//Outlet element evaluation
	if(domain[x].flag==1)
	{
		domain[x].stateVar[y]=initial[y];
	}		
}

void ausmplus(float *initial,float timesteps, float deltat)
{
	//GPU variables
	cell *d_domain;
	float *d_node,*d_boundary,*d_initial;
	float *d_R,*d_k,*d_gammma,*d_mu;

	//Store the values of nodes for faster access
	float *nodes=new float[26233*3];

	//Values of boundary elements
	float *boundary=new float[2200*2];

	//Alocate memory to domain
	cell *domain=new cell[50266];

	cout<<"Allocation on the host PC : done"<<endl;
	cout<<endl;

	//Open mesh files
	fstream myfile1,myfile2,myfile3;
	myfile1.open("Elements.txt",ios::in);
	myfile2.open("Nodes.txt",ios::in);
	myfile3.open("boundary.txt",ios::in);

	//Fill the array nodes for access in GPU
	if(myfile2.is_open())
	{
		for(int i=0;i<26233*3;i+=3)
		{
			myfile2>>nodes[i]>>nodes[i+1]>>nodes[i+2];
		}
	}
	else
	{
		cout<<"Could not open Nodes.txt"<<endl;
	}

	//Fill the array boundary for access in GPU
	if(myfile3.is_open())
	{
		for(int i=0;i<2200*2;i+=2)
		{
			myfile3>>boundary[i]>>boundary[i+1];	
		}
	}
	else
	{
		cout<<"Could not open boundary.txt"<<endl;
	}
	//Feed the file just once for access in GPU
	if(myfile1.is_open())
	{
		for(int i=0;i<50266;i++)
		{	
			domain[i]=cell(initial);
			myfile1>>domain[i].nodes[0][2]>>domain[i].nodes[1][2]>>domain[i].nodes[2][2];
		}
	}
	else
	{
		cout<<"Could not open Elements.txt"<<endl;	
	}
	myfile1.close();
	myfile2.close();
	myfile3.close();

	cout<<"Initialisation : done"<<endl;
	cout<<endl;

	hipStream_t stream1, stream2,stream3;
	hipStreamCreate(&stream1);
	hipStreamCreate(&stream2);
	hipStreamCreate(&stream3);

	hipMalloc((void **)&d_domain,50266*sizeof(cell));
	hipMalloc((void **)&d_node,26233*3*sizeof(float));
	hipMalloc((void **)&d_boundary,2200*2*sizeof(float));
	hipMalloc((void **)&d_initial,4*sizeof(float));
	hipMalloc((void **)&d_R,sizeof(float));
	hipMalloc((void **)&d_k,sizeof(float));
	hipMalloc((void **)&d_gammma,sizeof(float));
	hipMalloc((void **)&d_mu,sizeof(float));

	cout<<"Allocation on the GPU : done"<<endl;
	cout<<endl;

	hipMemcpyAsync(d_domain,&domain[0],50266*sizeof(cell),hipMemcpyHostToDevice,stream1);
	hipMemcpyAsync(d_node,&nodes[0],26233*3*sizeof(float),hipMemcpyHostToDevice,stream2);
	hipMemcpyAsync(d_boundary,&boundary[0],2200*2*sizeof(float),hipMemcpyHostToDevice,stream3);
	hipMemcpyAsync(d_initial,&initial[0],4*sizeof(float),hipMemcpyHostToDevice,stream3);
	hipMemcpyAsync(d_R,&R,sizeof(float),hipMemcpyHostToDevice,stream3);
	hipMemcpyAsync(d_k,&k,sizeof(float),hipMemcpyHostToDevice,stream3);
	hipMemcpyAsync(d_gammma,&gammma,sizeof(float),hipMemcpyHostToDevice,stream3);
	hipMemcpyAsync(d_mu,&mu,sizeof(float),hipMemcpyHostToDevice,stream3);

	cout<<"Memory copy on the GPU : done"<<endl;
	cout<<endl;

	set_nodes<<<50266,3>>>(d_node,d_domain,d_boundary,d_initial,d_gammma);
	set_neighbour<<<50266,3>>>(d_domain);
	calculate_norm<<<50266,3>>>(d_domain);
	read_values<<<50266,3>>>(d_domain);

	hipDeviceSynchronize();

	cout<<"Initialisation on the GPU : done"<<endl;
	cout<<endl;

	//Euler first order method

	for (float t = 0; t < timesteps*deltat; t+=deltat)
	{
		pressureFlux<<<50266,3,0,stream1>>>(d_domain,d_R,d_gammma);
		convectiveflux<<<50266,3,0,stream2>>>(d_domain,d_R,d_gammma);
		diffusiveFlux<<<50266,3,0,stream3>>>(d_domain,d_R,d_gammma,d_mu,300,d_k);
		hipDeviceSynchronize();
		if((int)(t/deltat)%1000000==0)
		{
			hipMemcpyAsync(&domain[0],d_domain,50266*sizeof(cell),hipMemcpyDeviceToHost,stream3);
			visual(domain,t);
		}
		evaluate<<<50266,4>>>(d_domain,deltat);
		//Boundary<<<50266,3,0,stream1>>>(d_domain,d_initial);
		hipDeviceSynchronize();
		cout<<"time = "<<t<<endl;
		cout<<endl;
		read_values<<<50266,3,0,stream1>>>(d_domain);
		hipDeviceSynchronize();
	}

	cout<<endl;
	hipMemcpy(&domain[0],d_domain,50266*sizeof(cell),hipMemcpyDeviceToHost);

	cout<<"Copying final values on the CPU from GPU : done"<<endl;
	cout<<endl;

	visual(domain,deltat*timesteps);
	hipStreamDestroy(stream1);
	hipStreamDestroy(stream2);
	hipStreamDestroy(stream3);	

	hipFree(d_initial);
	hipFree(d_boundary);
	hipFree(d_node);
	hipFree(d_domain);

	delete[] nodes;
	delete[] boundary;
	delete[] domain;
}
