#include "hip/hip_runtime.h"
#include "ausmPlus.h"
#include <math.h>
#include <algorithm>
#include <stdio.h>

__global__ void pressureFlux(cell *domain, double *R, double *gammma)
{
	int y=threadIdx.x;
	int x=blockIdx.x;
	int ourFlag=(int)domain[x].flag;
	if(ourFlag==0 || ourFlag==4)
	{

		//Calculating the critical speed of sound for all the four sides/faces and the element itself
		double a_s[2];

		//Element
		a_s[0]=sqrt(2*(gammma[0]-1)/(gammma[0]+1)*(domain[x].stateVar[3]+(gammma[0]-1)*(domain[x].stateVar[3]\
			-0.5*(pow(domain[x].stateVar[1],2)+pow(domain[x].stateVar[2],2))/domain[x].stateVar[0]))/domain[x].stateVar[0]);
		//Side/face
		a_s[1]=sqrt(2*(gammma[0]-1)/(gammma[0]+1)*(domain[x].temp_var[y][3]+(gammma[0]-1)*(domain[x].temp_var[y][3]\
			-0.5*(pow(domain[x].temp_var[y][1],2)+pow(domain[x].temp_var[y][2],2))/domain[x].temp_var[y][0]))/domain[x].temp_var[y][0]);
		
		//speed for the boundary calculation
		a_s[0]=pow(a_s[0],2)/max(a_s[0],abs(sqrt(pow(domain[x].stateVar[1]/domain[x].stateVar[0],2)+pow(domain[x].stateVar[2]/domain[x].stateVar[0],2))));
		a_s[1]=pow(a_s[1],2)/max(a_s[1],abs(sqrt(pow(domain[x].temp_var[y][1]/domain[x].temp_var[y][0],2)+pow(domain[x].temp_var[y][2]/domain[x].temp_var[y][0],2))));

		//Speed of sound at facial interface
		double a_mid=min(a_s[0],a_s[1]);
		//Mach number of incoming and outgoing waves
		double machplus=(domain[x].stateVar[1]/domain[x].stateVar[0]*domain[x].norms[y][0]+domain[x].stateVar[2]/domain[x].stateVar[0]*domain[x].norms[y][1])/a_mid;
		double machminus=(domain[x].temp_var[y][1]/domain[x].temp_var[y][0]*domain[x].norms[y][0]+domain[x].temp_var[y][2]/domain[x].temp_var[y][0]*domain[x].norms[y][1])/a_mid;
		
		//Pressure Fluxes
		double pressplus=(gammma[0]-1)*(domain[x].stateVar[3]-0.5*(pow(domain[x].stateVar[1],2)+pow(domain[x].stateVar[2],2))/domain[x].stateVar[0]);
		double presminus=(gammma[0]-1)*(domain[x].temp_var[y][3]-0.5*(pow(domain[x].temp_var[y][1],2)+pow(domain[x].temp_var[y][2],2))/domain[x].temp_var[y][0]);

		double plus,minus;
		if(abs(machplus)>=1)
			plus=0.5*(1+machplus/abs(machplus));
		else
			plus=0.25*pow((machplus+1),2)*(2-machplus)+3/16*machplus*pow((pow(machplus,2)-1),2);
		if(abs(machminus)>=1)
			minus=0.5*(1-machminus/abs(machminus));
		else
			minus=0.25*pow((machminus-1),2)*(2+machminus)-3/16*machminus*pow((pow(machminus,2)-1),2);

		domain[x].presflux[y][0]=(pressplus*plus+presminus*minus)*domain[x].norms[y][0]*sqrt(pow(domain[x].nodes[(y+1)%4][0]-domain[x].nodes[y][0],2)\
		+pow(domain[x].nodes[(y+1)%4][1]-domain[x].nodes[y][1],2));
		domain[x].presflux[y][1]=(pressplus*plus+presminus*minus)*domain[x].norms[y][1]*sqrt(pow(domain[x].nodes[(y+1)%4][0]-domain[x].nodes[y][0],2)\
		+pow(domain[x].nodes[(y+1)%4][1]-domain[x].nodes[y][1],2));

		//if(domain[x].flag==4)
		//	printf("%lf %lf %d %d\n",domain[x].presflux[y][0],domain[x].presflux[y][1],x,y );
	}
}
