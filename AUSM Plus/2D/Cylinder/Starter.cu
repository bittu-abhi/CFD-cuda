#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cmath>
#include "ausmPlus.h"

using namespace std;

double gammma;
double mu;
double k;
double R;	

int main()
{
	gammma=1.4;
	mu=pow(1.798,-5);
	k=0.0251;
	R=286.9;
	
	double initial[4];
	//Rho
	initial[0]=1.225;
	//Rho*U
	initial[1]=0.1*initial[0];
	//Rho *V
	initial[2]=0;
	//Rho*E, E is the internal energy including the kinetic energy(i.e. total intenal energy)
	initial[3]=101325/(gammma-1)+0.5*(pow(initial[1],2)+pow(initial[2],2))/initial[0];
	//Time steps and delta_t
	double timesteps=100;
	double deltat=0.000001;
	ausmplus(initial,timesteps,deltat);

	return 0;
}
