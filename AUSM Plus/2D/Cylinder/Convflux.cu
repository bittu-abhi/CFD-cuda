#include "hip/hip_runtime.h"
#include "ausmPlus.h"
#include <math.h>
#include <stdio.h>


__global__ void convectiveflux(cell *domain, float *R, float *gammma)
{
	int y=threadIdx.x;
	int x=blockIdx.x;
	int ourFlag=(int)domain[x].flag;
	if(ourFlag==0 || ourFlag==4 || ourFlag==2)
	{
		//Calculating the critical speed of sound for all the four sides/faces and the element itself
		float a_s[2];

		//Element
		a_s[0]=sqrt(2*(gammma[0]-1)/(gammma[0]+1)*(domain[x].stateVar[3]+(gammma[0]-1)*(domain[x].stateVar[3]\
			-0.5*(powf(domain[x].stateVar[1],2)+powf(domain[x].stateVar[2],2))/domain[x].stateVar[0]))/domain[x].stateVar[0]);
		//Side/face
		a_s[1]=sqrt(2*(gammma[0]-1)/(gammma[0]+1)*(domain[x].temp_var[y][3]+(gammma[0]-1)*(domain[x].temp_var[y][3]\
			-0.5*(powf(domain[x].temp_var[y][1],2)+powf(domain[x].temp_var[y][2],2))/domain[x].temp_var[y][0]))/domain[x].temp_var[y][0]);

		//speed for the boundary calculation
		a_s[0]=powf(a_s[0],2)/max(a_s[0],abs(sqrt(powf(domain[x].stateVar[1]/domain[x].stateVar[0],2)+powf(domain[x].stateVar[2]/domain[x].stateVar[0],2))));
		a_s[1]=powf(a_s[1],2)/max(a_s[1],abs(sqrt(powf(domain[x].temp_var[y][1]/domain[x].temp_var[y][0],2)+powf(domain[x].temp_var[y][2]/domain[x].temp_var[y][0],2))));

		//Speed of sound at facial interface
		float a_mid=min(a_s[0],a_s[1]);

		//Pressure
		float pressplus=(*gammma-1)*(domain[x].stateVar[3]-0.5*(powf(domain[x].stateVar[1],2)+powf(domain[x].stateVar[2],2))/domain[x].stateVar[0]);
		float pressminus=(*gammma-1)*(domain[x].temp_var[y][3]-0.5*(powf(domain[x].temp_var[y][1],2)+powf(domain[x].temp_var[y][2],2))/domain[x].temp_var[y][0]);

		//Machnumber of contravarient velocity(V=u*nx+v*ny)
		float mach_one=(domain[x].stateVar[1]/domain[x].stateVar[0]*domain[x].norms[y][0]+domain[x].stateVar[2]/domain[x].stateVar[0]*domain[x].norms[y][1])/a_mid;
		float mach_two=(domain[x].temp_var[y][1]/domain[x].temp_var[y][0]*domain[x].norms[y][0]+domain[x].temp_var[y][2]/domain[x].temp_var[y][0]*domain[x].norms[y][1])/a_mid;

		float split_mach_one,split_mach_two;
		if(abs(mach_one)>=1)
			split_mach_one=0.5*(mach_one+abs(mach_one));
		else
			split_mach_one=(0.25*powf(mach_one+1,2))+((1.0/8.0)*(powf(powf(mach_one,2.0)-1.0,2.0)));

		if(abs(mach_two)>=1)
			split_mach_two=0.5*(mach_two-abs(mach_two));
		else
			split_mach_two=-0.25*powf(mach_two-1.0,2.0)-1.0/8.0*powf(powf(mach_two,2.0)-1.0,2.0);
	
			float split_mach=split_mach_one+split_mach_two;
		
		for (int i = 0; i < 4; ++i)
		{
			domain[x].convflux[y][i]=a_mid*(0.5*(split_mach+abs(split_mach))*domain[x].stateVar[i]+0.5*(split_mach-abs(split_mach))\
				*domain[x].temp_var[y][i])*sqrt(powf(domain[x].nodes[y][0]-domain[x].nodes[(y+1)%3][0],2)+powf(domain[x].nodes[y][1]-domain[x].nodes[(y+1)%3][1],2));
		}
		domain[x].convflux[y][3]+=a_mid*(0.5*(split_mach+abs(split_mach))*pressplus+0.5*(split_mach-abs(split_mach))*pressminus)\
		*sqrt(powf(domain[x].nodes[y][0]-domain[x].nodes[(y+1)%3][0],2)+powf(domain[x].nodes[y][1]-domain[x].nodes[(y+1)%3][1],2));

		if(abs((1.0/3.0)*(domain[x].nodes[0][0]+domain[x].nodes[1][0]+domain[x].nodes[2][0])+0.524158619)<0.001 && abs((1.0/3.0)*(domain[x].nodes[0][1]+domain[x].nodes[1][1]+domain[x].nodes[2][1])-0.8526501336)<0.001)
			printf("convective %5.14lf %5.14lf %5.14lf %5.14lf	 %d %d %d\n",domain[x].convflux[y][0],domain[x].convflux[y][1],domain[x].convflux[y][2],domain[x].convflux[y][3],domain[x].flag,x+1,y);
	}
}
