#include "hip/hip_runtime.h"
#include "ausmPlus.h"
#include <hip/hip_runtime_api.h>
#include <math.h>

__global__ void set_nodes(double *node, cell *domain, double *boundary)
{
	int x=blockIdx.x;
	int y=threadIdx.x;
	int flag1=0;
	int temp=3*((int)(domain[x].nodes[y][2])-1);
	domain[x].nodes[y][1]=node[temp+1];
	domain[x].nodes[y][0]=node[temp];
	if(domain[x].nodes[y][2]>126 && domain[x].nodes[y][2]<176)
	{
		domain[x].flag=1;	
	}
	else if(domain[x].nodes[y][2]>=25402 && domain[x].nodes[y][2]<=25452)
	{
		domain[x].flag=2;
	}
	for(int i=0;i<600*2;i++)
	{
		if(domain[x].nodes[0][2]==boundary[i] || domain[x].nodes[1][2]==boundary[i] || domain[x].nodes[2][2]==boundary[i] || domain[x].nodes[3][2]==boundary[i])
		{
			flag1=1;
			break;
		}
	}
	if(flag1==0)
		domain[x].flag=0;

	for (int i = 0; i < 4; ++i)
	{
		if(abs(domain[x].nodes[i][0]-(20+sqrt(4-pow(domain[x].nodes[i][1]-15,2))))<0.005 || abs(domain[x].nodes[i][0]-(20-sqrt(4-pow(domain[x].nodes[i][1]-15,2))))<0.005)
			domain[x].flag=4;
	}
	if(flag1==1 && domain[x].flag!=1 && domain[x].flag!=2 && domain[x].flag!=4 && domain[x].flag!=0)
			domain[x].flag=3;
}

__global__ void set_neighbour(cell *domain)
{
	int x=blockIdx.x;
	int y=threadIdx.x;
	int i,flag1=0,flag2=0;
	for (i = 0; i < 25000; i+=1)
	{
		for (int j = 0; j < 4; j+=1)
		{
			if(domain[i].nodes[j][0]==domain[x].nodes[y][0] && i!=x && domain[i].nodes[j][1]==domain[x].nodes[y][1])
				flag1=1;
			if( domain[i].nodes[j][0]==domain[x].nodes[(y+1)%4][0] && i!=x && domain[i].nodes[j][1]==domain[x].nodes[(y+1)%4][1])
				flag2=1;
		}
		if(flag1==1 && flag2==1)
		{	
			domain[x].face[y]=i;
			break;
		}
		flag1=0;
		flag2=0;
	}
}

__global__ void calculate_norm(cell *domain)
{
	int x=blockIdx.x;
	int y=threadIdx.x;

	//Now to determine if the normal is pointing outward, and if not, then change accordingly
	double cen_cord[2];
	cen_cord[0]=0.25*(domain[x].nodes[0][0]+domain[x].nodes[1][0]+domain[x].nodes[2][0]+domain[x].nodes[3][0]);
	cen_cord[1]=0.25*(domain[x].nodes[0][1]+domain[x].nodes[1][1]+domain[x].nodes[2][1]+domain[x].nodes[3][1]);

	//construct the face
	double m,c;
	//if(domain[x].nodes[(y+1)%4][0]-domain[x].nodes[y][0]!=0)
		m=(domain[x].nodes[(y+1)%4][1]-domain[x].nodes[y][1])/(domain[x].nodes[(y+1)%4][0]-domain[x].nodes[y][0]);

	c=domain[x].nodes[y][1]-m*domain[x].nodes[y][0];

	//A perpendicular line passing through the centre of the element
	if(m!=0 && !isinf(m))
	{
		double req_m=-1/m;
		double req_c=cen_cord[1]-req_m*cen_cord[0];

		//Intersection of this line with the face would give a point on the face. Now using this point as (x1,y2), we would
		//always get a vector pointing outward from the face,regardless of the way the nodes are number(clockwise or anticlockwise)
		double req_x=(c-req_c)/(req_m-m);
		double req_y=m*req_x+c;
		
		double dino=sqrt(pow(req_x-cen_cord[0],2)+pow(req_y-cen_cord[1],2));

		domain[x].norms[y][0]=(req_x-cen_cord[0])/dino;
		domain[x].norms[y][1]=(req_y-cen_cord[1])/dino;
	}
	else if(m==0)
	{
		domain[x].norms[y][0]=0;
		if(domain[x].nodes[(y+1)%4][1]<cen_cord[1])
			domain[x].norms[y][1]=-1;
		else
			domain[x].norms[y][1]=1;
	}
	else
	{
		domain[x].norms[y][1]=0;
		if(domain[x].nodes[(y+1)%4][0]<cen_cord[0])
			domain[x].norms[y][0]=-1;
		else
			domain[x].norms[y][0]=1;
	}
}
