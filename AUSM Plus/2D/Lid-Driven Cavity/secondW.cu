#include "hip/hip_runtime.h"
#include "lid.h"
#include <stdio.h>

__global__ void secondW(point *pt, int points, double delta, int flagXY )
{
	int y_level=blockIdx.x;
	int x = threadIdx.x;

	if(x!=0 && y_level!=0 && x!=points-1 && y_level!=points-1)
	{
		pt[(x+y_level*points)].delW[flagXY] = (pt[(flagXY==0)?(x+1+y_level*points):((y_level+1)*points+x)].w-pt[(flagXY==0)?(x-1+y_level*points):((y_level-1)*points+x)].w)/(2*delta);

		pt[(x+y_level*points)].delpsi[flagXY] = (pt[(flagXY==0)?(x+1+y_level*points):((y_level+1)*points+x)].psi-pt[(flagXY==0)?(x-1+y_level*points):((y_level-1)*points+x)].psi)/(2*delta);


		pt[(x+y_level*points)].delW2[flagXY]=(pt[(flagXY==0)?(x-1+points*y_level):((y_level-1)*points+x)].w-2*pt[(flagXY==0)?(x+points*y_level):(y_level*points+x)].w+pt[(flagXY==0)?(x+1+points*y_level):((y_level+1)*points+x)].w)/(delta*delta);
	}
	if(x==points-1)
	{

		if(y_level!=(points-1) && y_level!=0)
		{	
			pt[(x+y_level*points)].delW[flagXY] = (flagXY==0)?((pt[(x+points*y_level)].w-pt[((x-1)+points*y_level)].w)/(delta)): (pt[((y_level+1)*points+x)].w-pt[((y_level-1)*points+x)].w)/(2*delta);

			pt[(x+y_level*points)].delpsi[flagXY] = (flagXY==0)?((pt[(x+points*y_level)].psi-pt[((x-1)+points*y_level)].psi)/(delta)): (pt[((y_level+1)*points+x)].psi-pt[((y_level
				-1)*points+x)].psi)/(2*delta);

			pt[(x+y_level*points)].delW2[flagXY] = (flagXY==0)?(2*(pt[(x+points*y_level)].w-5*pt[((x-1)+points*y_level)].w+4*pt[((x-2)+points*y_level)].w-pt[((x-3)+points*y_level)].w)/(delta*delta)): (pt[((y_level-1)*points+x)].w-2*pt[(y_level*points+x)].w+pt[((y_level+1)*points+x)].w)/(delta*delta);
		}
	}
	if(x==0)
	{
		if(y_level!=(points-1) && y_level!=0)
		{
			pt[x+y_level*points].delW[flagXY] = (flagXY==0)?((pt[(x+1+points*y_level)].w-pt[((x)+points*y_level)].w)/(delta)): (pt[((y_level+1)*points+x)].w-pt[((y_level-1)*points+x)].w)/(2*delta);

			pt[(x+y_level*points)].delpsi[flagXY] = (flagXY==0)?((pt[(x+1+points*y_level)].psi-pt[((x)+points*y_level)].psi)/(delta)): (pt[((y_level+1)*points+x)].psi-pt[((y_level-1)*points+x)].psi)/(2*delta);

			pt[(x+y_level*points)].delW2[flagXY] = (flagXY==0)?(2*(pt[(x+points*y_level)].w-5*pt[((x+1)+points*y_level)].w+4*pt[((x+2)+points*y_level)].w-pt[((x+3)+points*y_level)].w)/(delta*delta)): (pt[((y_level-1)*points+x)].w-2*pt[(y_level*points+x)].w+pt[((y_level+1)*points+x)].w)/(delta*delta);
		}
	}
	if(y_level==0)
	{
		if(x!=(points-1) && x!=0)
		{
			pt[(x+y_level*points)].delW[flagXY] = (flagXY==0)?((pt[(x+1+points*y_level)].w-pt[((x-1)+points*y_level)].w)/(2*delta)): (pt[((y_level+1)*points+x)].w-pt[((y_level)*points+x)].w)/(delta);

			pt[(x+y_level*points)].delpsi[flagXY] = (flagXY==0)?((pt[(x+1+points*y_level)].psi-pt[((x-1)+points*y_level)].psi)/(2*delta)): (pt[((y_level+1)*points+x)].psi-pt[((y_level)*points+x)].psi)/(delta);

			pt[(x+y_level*points)].delW2[flagXY] = (flagXY==0)?((pt[(x+1+points*y_level)].w+pt[((x-1)+points*y_level)].w-2*pt[((x)+points*y_level)].w)/(delta*delta)): (2*pt[((y_level)*points+x)].w-5*pt[((y_level+1)*points+x)].w+4*pt[((y_level+2)*points+x)].w-pt[((y_level+3)*points+x)].w)/(delta*delta);
		}	
	}
	if(y_level==(points-1))
	{
		if(x!=(points-1) && x!=0)
		{
			pt[(x+y_level*points)].delW[flagXY] = (flagXY==0)?((pt[(x+1+points*y_level)].w-pt[((x-1)+points*y_level)].w)/(2*delta)): (pt[((y_level)*points+x)].w-pt[((y_level-1)*points+x)].w)/(delta);

			pt[(x+y_level*points)].delpsi[flagXY] = (flagXY==0)?((pt[(x+1+points*y_level)].psi-pt[((x-1)+points*y_level)].psi)/(2*delta)): (pt[((y_level)*points+x)].psi-pt[((y_level-1)*points+x)].psi)/(delta);

			pt[(x+y_level*points)].delW2[flagXY] = (flagXY==0)?((pt[(x+1+points*y_level)].w+pt[((x-1)+points*y_level)].w-2*pt[((x)+points*y_level)].w)/(delta*delta)): (2*pt[((y_level)*points+x)].w-5*pt[((y_level-1)*points+x)].w+4*pt[((y_level-2)*points+x)].w-pt[((y_level-3)*points+x)].w)/(delta*delta);
		}
	}

	//Four Cornor points
	if(y_level==(points-1) && x==points-1)
	{
		pt[(x+y_level*points)].delW[flagXY] = (flagXY==0)?((pt[(x+points*y_level)].w-pt[((x-1)+points*y_level)].w)/(delta)): (pt[((y_level)*points+x)].w-pt[((y_level-1)*points+x)].w)/(delta);

		pt[(x+y_level*points)].delpsi[flagXY] = (flagXY==0)?((pt[(x+points*y_level)].psi-pt[((x-1)+points*y_level)].psi)/(delta)): (pt[((y_level)*points+x)].psi-pt[((y_level-1)*points+x)].psi)/(delta);

		pt[(x+y_level*points)].delW2[flagXY] = (flagXY==0)?(2*(pt[(x+points*y_level)].w-5*pt[((x-1)+points*y_level)].w+4*pt[((x-2)+points*y_level)].w-pt[((x-3)+points*y_level)].w)/(delta*delta)): (2*pt[((y_level)*points+x)].w-5*pt[((y_level-1)*points+x)].w+4*pt[((y_level-2)*points+x)].w-pt[((y_level-3)*points+x)].w)/(delta*delta);
	}
	if(y_level==(points-1) && x==0)
	{
		pt[(x+y_level*points)].delW[flagXY] = (flagXY==0)?((pt[((x+1)+points*y_level)].w-pt[((x)+points*y_level)].w)/(delta)): (pt[((y_level)*points+x)].w-pt[((y_level-1)*points+x)].w)/(delta);

		pt[(x+y_level*points)].delpsi[flagXY] = (flagXY==0)?((pt[((x+1)+points*y_level)].psi-pt[((x+0)+points*y_level)].psi)/(delta)): (pt[((y_level)*points+x)].psi-pt[((y_level-1)*points+x)].psi)/(delta);

		pt[(x+y_level*points)].delW2[flagXY] = (flagXY==0)?(2*(pt[(x+points*y_level)].w-5*pt[((x+1)+points*y_level)].w+4*pt[((x+2)+points*y_level)].w-pt[((x+3)+points*y_level)].w)/(delta*delta)): (2*pt[((y_level)*points+x)].w-5*pt[((y_level-1)*points+x)].w+4*pt[((y_level-2)*points+x)].w-pt[((y_level-3)*points+x)].w)/(delta*delta);
	}
	if(y_level==0 && x==(points-1))
	{
		pt[(x+y_level*points)].delW[flagXY] = (flagXY==0)?((pt[(x+points*y_level)].w-pt[((x-1)+points*y_level)].w)/(delta)): (pt[((y_level+1)*points+x)].w-pt[((y_level)*points+x)].w)/(delta);

		pt[(x+y_level*points)].delpsi[flagXY] = (flagXY==0)?((pt[(x+points*y_level)].psi-pt[((x-1)+points*y_level)].psi)/(delta)): (pt[((y_level+1)*points+x)].psi-pt[((y_level)*points+x)].psi)/(delta);

		pt[(x+y_level*points)].delW2[flagXY] = (flagXY==0)?(2*(pt[(x+points*y_level)].w-5*pt[((x-1)+points*y_level)].w+4*pt[((x-2)+points*y_level)].w-pt[((x-3	)+points*y_level)].w)/(delta*delta)): (2*pt[((y_level)*points+x)].w-5*pt[((y_level+1)*points+x)].w+4*pt[((y_level+2)*points+x)].w-pt[((y_level+3)*points+x)].w)/(delta*delta);
	}
	if(y_level==0 && x==0)
	{
		pt[(x+y_level*points)].delW[flagXY] = (flagXY==0)?((pt[((x+1)+points*y_level)].w-pt[((x)+points*y_level)].w)/(delta)): (pt[((y_level+1)*points+x)].w-pt[((y_level)*points+x)].w)/(delta);

		pt[(x+y_level*points)].delpsi[flagXY] = (flagXY==0)?((pt[((x+1)+points*y_level)].psi-pt[((x)+points*y_level)].psi)/(delta)): (pt[((y_level+1)*points+x)].psi-pt[((y_level)*points+x)].psi)/(delta);

		pt[(x+y_level*points)].delW2[flagXY] = (flagXY==0)?(2*(pt[(x+points*y_level)].w-5*pt[((x+1)+points*y_level)].w+4*pt[((x+2)+points*y_level)].w-pt[((x+3)+points*y_level)].w)/(delta*delta)): (2*pt[((y_level)*points+x)].w-5*pt[((y_level+1)*points+x)].w+4*pt[((y_level+2)*points+x)].w-pt[((y_level+3)*points+x)].w)/(delta*delta);
	}

	//if(y_level==(points-1) && x==(points-1))
	//		printf("product=%5.14lf, delWX=%5.14lf, delWY=%5.14lf, flagXY=%d  %d %d\n",(flagXY==0)?((pt[(x+points*y_level)].w-pt[((x-1)+points*y_level)].w)/(delta)): (pt[((y_level)*points+x)].w-pt[((y_level-1)*points+x)].w)/(delta),pt[(x+points*y_level)].delW[0],pt[(x+points*y_level)].delW[1],flagXY,x,y_level);
}