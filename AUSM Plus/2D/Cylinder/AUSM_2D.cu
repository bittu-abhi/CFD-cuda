#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <omp.h>
#include "ausmPlus.h"

using namespace std;

cell::cell(double *state)
{
	omp_set_nested(1);
	omp_set_num_threads(4);
	#pragma omp parallel for
	for(int i=0;i<4;i++)
	{
		stateVar[i]=state[i];
	}
}

cell::cell(){}

__global__ void evaluate(cell *domain,double deltat)
{
	int x=blockIdx.x;
	int y=threadIdx.x;
	double vol=0.5*(domain[x].nodes[0][0]-domain[x].nodes[2][0])*(domain[x].nodes[1][1]-domain[x].nodes[3][1])+\
	0.5*(domain[x].nodes[3][0]-domain[x].nodes[1][0])*(domain[x].nodes[0][1]-domain[x].nodes[2][1]);
	if(domain[x].flag==0 || domain[x].flag==4)
	{
		domain[x].stateVar[y]=domain[x].stateVar[y]-(domain[x].convflux[0][y]+domain[x].convflux[1][y]+domain[x].convflux[2][y]+domain[x].convflux[3][y]\
			-(domain[x].diffflux[0][y]+domain[x].diffflux[1][y]+domain[x].diffflux[2][y]+domain[x].diffflux[3][y]))/vol*deltat;
		if(y==1)
			domain[x].stateVar[y]+=(domain[x].presflux[0][0]+domain[x].presflux[1][0]+domain[x].presflux[2][0]+domain[x].presflux[3][0])/vol*deltat;
		if(y==2)
			domain[x].stateVar[y]+=(domain[x].presflux[0][1]+domain[x].presflux[1][1]+domain[x].presflux[2][1]+domain[x].presflux[3][1])/vol*deltat;
	}
}

__global__ void Boundary(cell *domain,double *initial)
{
	int x=blockIdx.x;
	int y=threadIdx.x;
	//Inlet element evaluation
	if(domain[x].flag==1)
	{
		domain[x].stateVar[y]=initial[y];
	}
	//Outlet element evaluation
	else if(domain[x].flag==2)
	{
		if(threadIdx.x>0)
		{
			domain[x].stateVar[y]=2*domain[x].stateVar[y]-domain[(int)domain[x].face[3][0]].stateVar[y];
		}
		else
			domain[x].stateVar[y]=initial[0];
	}
	//Farfield element evaluation
	else if(domain[x].flag==3)
	{
		/* nothing to change :-) */
	}
}

void ausmplus(double *initial,double timesteps, double deltat)
{
	//OpenMP flags
	omp_set_nested(1);
	omp_set_num_threads(2*omp_get_num_procs());

	//GPU variables
	cell *d_domain;
	double *d_node,*d_boundary,*d_initial;
	double *d_R,*d_k,*d_gammma,*d_mu;

	//Store the values of nodes for faster access
	double *nodes=new double[25452*3];

	//Values of boundary elements
	double *boundary=new double[599*2];

	//Alocate memory to domain
	cell *domain=new cell[25000];

	cout<<"Allocation on the host PC : done"<<endl;
	cout<<endl;

	//Open mesh files
	fstream myfile1,myfile2,myfile3;
	myfile1.open("Elements.txt",ios::in);
	myfile2.open("Nodes.txt",ios::in);
	myfile3.open("boundary.txt",ios::in);

	//Fill the array nodes for access in GPU
	for(int i=0;i<25452*3;i+=3)
	{
		myfile2>>nodes[i]>>nodes[i+1]>>nodes[i+2];
	}

	//Fill the array boundary for access in GPU
	for(int i=0;i<599*2;i+=2)
	{
		myfile3>>boundary[i]>>boundary[i+1];
	}

	//Feed the file just once for access in GPU
	for(int i=0;i<25000;i++)
	{
		domain[i]=cell(initial);
		myfile1>>domain[i].nodes[0][2]>>domain[i].nodes[1][2]>>domain[i].nodes[2][2]>>domain[i].nodes[3][2];
	}
	myfile1.close();
	myfile2.close();
	myfile3.close();

	cout<<"Initialisation : done"<<endl;
	cout<<endl;

	hipStream_t stream1, stream2,stream3,stream4;
	hipStreamCreate(&stream1);
	hipStreamCreate(&stream2);
	hipStreamCreate(&stream3);
	hipStreamCreateWithFlags(&stream4,hipStreamNonBlocking);
	hipMalloc((void **)&d_domain,25000*sizeof(cell));
	hipMalloc((void **)&d_node,25452*3*sizeof(double));
	hipMalloc((void **)&d_boundary,599*2*sizeof(double));
	hipMalloc((void **)&d_initial,4*sizeof(double));
	hipMalloc((void **)&d_R,sizeof(double));
	hipMalloc((void **)&d_k,sizeof(double));
	hipMalloc((void **)&d_gammma,sizeof(double));
	hipMalloc((void **)&d_mu,sizeof(double));

	cout<<"Allocation on the GPU : done"<<endl;
	cout<<endl;

	hipMemcpyAsync(d_domain,&domain[0],25000*sizeof(cell),hipMemcpyHostToDevice,stream1);
	hipMemcpyAsync(d_node,&nodes[0],25452*3*sizeof(double),hipMemcpyHostToDevice,stream2);
	hipMemcpyAsync(d_boundary,&boundary[0],599*2*sizeof(double),hipMemcpyHostToDevice,stream3);
	hipMemcpyAsync(d_initial,&initial[0],4*sizeof(double),hipMemcpyHostToDevice,stream3);
	hipMemcpyAsync(d_R,&R,sizeof(double),hipMemcpyHostToDevice,stream3);
	hipMemcpyAsync(d_k,&k,sizeof(double),hipMemcpyHostToDevice,stream3);
	hipMemcpyAsync(d_gammma,&gammma,sizeof(double),hipMemcpyHostToDevice,stream3);
	hipMemcpyAsync(d_mu,&mu,sizeof(double),hipMemcpyHostToDevice,stream3);

	cout<<"Memory copy on the GPU : done"<<endl;
	cout<<endl;

	set_nodes<<<25000,4>>>(d_node,d_domain,d_boundary);
	set_neighbour<<<25000,4>>>(d_domain);
	
	hipDeviceSynchronize();

	cout<<"Initialisation on the GPU : done"<<endl;
	cout<<endl;

	//Euler first order method
	for (double t = 0; t < timesteps*deltat; t+=deltat)
	{
		pressureFlux<<<25000,4,0,stream1>>>(d_domain,d_R,d_gammma);
		convectiveflux<<<25000,4,0,stream2>>>(d_domain,d_R,d_gammma);
		diffusiveFlux<<<25000,4,0,stream3>>>(d_domain,d_R,d_gammma,d_mu,300,d_k);
		hipDeviceSynchronize();
		evaluate<<<25000,4>>>(d_domain,deltat);
		Boundary<<<25000,4,0,stream4>>>(d_domain,d_initial);
		hipDeviceSynchronize();
		cout<<"time = "<<t<<endl;
	}

	cout<<endl;
	hipMemcpy(&domain[0],d_domain,25000*sizeof(cell),hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	cout<<"Copying final values on the CPU from GPU : done"<<endl;
	cout<<endl;

	visual(domain);
	hipStreamDestroy(stream1);
	hipStreamDestroy(stream2);
	hipStreamDestroy(stream3);	
	hipStreamDestroy(stream4);	
	hipFree(d_node);
	hipFree(d_domain);

	delete[] nodes;
	delete[] boundary;

/*
	//For verification of the correct neighbours and nodes
	for (int i = 0; i < 25000; ++i)
	{
		//cout<<domain[i].diffflux[0][1]<<","<<domain[i].diffflux[1][1]<<","<<domain[i].diffflux[2][1]<<","<<domain[i].diffflux[3][1]<<endl;
		cout<<"("<<domain[i].nodes[0][0]<<","<<domain[i].nodes[0][1]<<")"<<","<<"("<<domain[i].nodes[1][0]<<","<<domain[i].nodes[1][1]<<")";
		cout<<"("<<domain[i].nodes[2][0]<<","<<domain[i].nodes[2][1]<<")"<<","<<"("<<domain[i].nodes[3][0]<<","<<domain[i].nodes[3][1]<<")"<<endl;
		//cout<<domain[i].face[0][0]<<","<<domain[i].face[1][0]<<","<<domain[i].face[2][0]<<","<<domain[i].face[3][0]<<"		"<<domain[i].flag<<endl;
		cout<<endl;
	} 
*/
	delete[] domain;
}
