#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <omp.h>
#include <algorithm>
#include <math.h>
#include "lid.h"

using namespace std;

point::point()
{}

int main()
{
	//number of points
	int points = 500;

	//dimension in x and y direction
	double x_dim = 1;
	double y_dim = 1;

	//delta values in different directions and time
	double deltax = x_dim/points;
	double deltay = y_dim/points;
	double deltat = 0.0001;

	//Total time steps
	double timesteps = 1000000;
	//Convergence cirterion
	double convergence = 0.000001;

	//total number of iterations to be performed
	int iter = 5000;

	//Velocity of the plate
	double u = 1;

	//density of the fluid
	double density = 1.225;
	
	//viscosity of the fluid
	double viscosity = 1.225*pow(10,-3);
	
	//Reynolds number calculated according to the dimensions of the cavity
	double reynold = max(x_dim,y_dim)*density*u/viscosity;

	cout<<"Reynolds number="<<reynold<<endl;

	LID(deltax, deltay, deltat, points, convergence, iter, reynold, timesteps);

	return 0;
}