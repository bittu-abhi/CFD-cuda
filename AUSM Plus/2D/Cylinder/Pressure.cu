#include "hip/hip_runtime.h"
#include "ausmPlus.h"
#include <math.h>
#include <algorithm>
#include <stdio.h>

__global__ void pressureFlux(cell *domain, double *R, double *gammma)
{
	int y=threadIdx.x;
	int x=blockIdx.x;
	int ourFlag=(int)domain[x].flag;
	if(ourFlag==0 || ourFlag==4)
	{


		//Calculating the critical speed of sound for all the four sides/faces and the element itself
		double a_s[2];

		//Element
		a_s[0]=sqrt(2*(gammma[0]-1)/(gammma[0]+1)*(domain[x].stateVar[3]+(gammma[0]-1)*(domain[x].stateVar[3]\
			-0.5*(pow(domain[x].stateVar[1],2)+pow(domain[x].stateVar[2],2))/domain[x].stateVar[0]))/domain[x].stateVar[0]);
		//Side/face
		a_s[1]=sqrt(2*(gammma[0]-1)/(gammma[0]+1)*(domain[x].temp_var[y][3]+(gammma[0]-1)*(domain[x].temp_var[y][3]\
			-0.5*(pow(domain[x].temp_var[y][1],2)+pow(domain[x].temp_var[y][2],2))/domain[x].temp_var[y][0]))/domain[x].temp_var[y][0]);
		
		//speed for the boundary calculation
		a_s[0]=pow(a_s[0],2)/max(a_s[0],abs(sqrt(pow(domain[x].stateVar[1]/domain[x].stateVar[0],2)+pow(domain[x].stateVar[2]/domain[x].stateVar[0],2))));
		a_s[1]=pow(a_s[1],2)/max(a_s[1],abs(sqrt(pow(domain[x].temp_var[y][1]/domain[x].temp_var[y][0],2)+pow(domain[x].temp_var[y][2]/domain[x].temp_var[y][0],2))));

		//Speed of sound at facial interface
		double a_mid=min(a_s[0],a_s[1]);

		//Mach number of incoming and outgoing waves
		double mach_one=(domain[x].stateVar[1]/domain[x].stateVar[0]*domain[x].norms[y][0]+domain[x].stateVar[2]/domain[x].stateVar[0]*domain[x].norms[y][1])/a_mid;
		double mach_two=(domain[x].temp_var[y][1]/domain[x].temp_var[y][0]*domain[x].norms[y][0]+domain[x].temp_var[y][2]/domain[x].temp_var[y][0]*domain[x].norms[y][1])/a_mid;
		
		//Pressure Fluxes
		double press_one=(*gammma-1)*(domain[x].stateVar[3]-0.5*(pow(domain[x].stateVar[1],2)+pow(domain[x].stateVar[2],2))/domain[x].stateVar[0]);
		double press_two=(*gammma-1)*(domain[x].temp_var[y][3]-0.5*(pow(domain[x].temp_var[y][1],2)+pow(domain[x].temp_var[y][2],2))/domain[x].temp_var[y][0]);

		double one,two;
		if(abs(mach_one)>=1)
			one=0.5*(1+mach_one/abs(mach_one));
		else
			one=0.25*pow((mach_one+1),2)*(2-mach_one)+3/16*mach_one*pow((pow(mach_one,2)-1),2);
	
		if(abs(mach_two)>=1)
			two=0.5*(1-mach_two/abs(mach_two));
		else
			two=0.25*pow((mach_two-1),2)*(2+mach_two)-3/16*mach_two*pow((pow(mach_two,2)-1),2);

		domain[x].presflux[y][0]=(press_one*one+press_two*two)*domain[x].norms[y][0]*sqrt(pow(domain[x].nodes[y][0]-domain[x].nodes[(y+1)%4][0],2)\
			+pow(domain[x].nodes[y][1]-domain[x].nodes[(y+1)%4][1],2));
		domain[x].presflux[y][1]=(press_one*one+press_two*two)*domain[x].norms[y][1]*sqrt(pow(domain[x].nodes[y][0]-domain[x].nodes[(y+1)%4][0],2)\
			+pow(domain[x].nodes[y][1]-domain[x].nodes[(y+1)%4][1],2));
	}
}
