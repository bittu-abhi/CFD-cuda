#include "hip/hip_runtime.h"
#include "ausmPlus.h"
#include <math.h>
#include <stdio.h>

__global__ void convectiveflux(cell *domain, double *R, double *gammma)
{
	int y=threadIdx.x;
	int x=blockIdx.x;
	int ourFlag=(int)domain[x].flag;
	if(ourFlag==0 || ourFlag==4)
	{
		//Calculating the critical speed of sound for all the four sides/faces and the element itself
		double a_s[2];

		//Element
		a_s[0]=sqrt(2*(gammma[0]-1)/(gammma[0]+1)*(domain[x].stateVar[3]+(gammma[0]-1)*(domain[x].stateVar[3]\
			-0.5*(pow(domain[x].stateVar[1],2)+pow(domain[x].stateVar[2],2))/domain[x].stateVar[0]))/domain[x].stateVar[0]);
		//Side/face
		a_s[1]=sqrt(2*(gammma[0]-1)/(gammma[0]+1)*(domain[x].temp_var[y][3]+(gammma[0]-1)*(domain[x].temp_var[y][3]\
			-0.5*(pow(domain[x].temp_var[y][1],2)+pow(domain[x].temp_var[y][2],2))/domain[x].temp_var[y][0]))/domain[x].temp_var[y][0]);

		//speed for the boundary calculation
		a_s[0]=pow(a_s[0],2)/max(a_s[0],abs(sqrt(pow(domain[x].stateVar[1]/domain[x].stateVar[0],2)+pow(domain[x].stateVar[2]/domain[x].stateVar[0],2))));
		a_s[1]=pow(a_s[1],2)/max(a_s[1],abs(sqrt(pow(domain[x].temp_var[y][1]/domain[x].temp_var[y][0],2)+pow(domain[x].temp_var[y][2]/domain[x].temp_var[y][0],2))));

		//Speed of sound at facial interface
		double a_mid=min(a_s[0],a_s[1]);

		//Pressure
		double press=domain[x].stateVar[3]+(*gammma-1)*(domain[x].stateVar[3]-0.5*(pow(domain[x].stateVar[1],2)+pow(domain[x].stateVar[2],2))/domain[x].stateVar[0]);

		//Machnumber of contravarient velocity(V=u*nx+v*ny)
		double machplus=(domain[x].stateVar[1]/domain[x].stateVar[0]*domain[x].norms[y][0]+domain[x].stateVar[2]/domain[x].stateVar[0]*domain[x].norms[y][1])/a_mid;
		double machminus=(domain[x].temp_var[y][1]/domain[x].temp_var[y][0]*domain[x].norms[y][0]+domain[x].temp_var[y][2]/domain[x].temp_var[y][0]*domain[x].norms[y][1])/a_mid;

		double split_mach_plus,split_mach_minus;

		if(abs(machplus)>=1)
			split_mach_plus=0.5*(machplus+abs(machplus));
		else
			split_mach_plus=0.5*pow(machplus+1.0,2.0)+1/8*pow(pow(machplus,2.0)-1.0,2.0);
		if(abs(machminus)>=1)
			split_mach_minus=0.5*(machminus-abs(machminus));
		else
			split_mach_minus=-0.5*pow(machminus-1.0,2.0)-1/8*pow(pow(machminus,2.0)-1.0,2.0);

		for (int i = 0; i < 4; ++i)
		{
			domain[x].convflux[y][i]=a_mid*(0.5*(split_mach_plus+abs(split_mach_plus))*domain[x].stateVar[i]+0.5*(split_mach_minus-abs(split_mach_minus))\
				*domain[x].temp_var[y][i])*sqrt(pow(domain[x].nodes[y][0]-domain[x].nodes[(y+1)%4][0],2)+pow(domain[x].nodes[y][1]-domain[x].nodes[(y+1)%4][1],2));
		}
		domain[x].convflux[y][3]+=a_mid*(0.5*(split_mach_plus+abs(split_mach_plus))*press+0.5*(split_mach_minus-abs(split_mach_minus))*press)\
		*sqrt(pow(domain[x].nodes[y][0]-domain[x].nodes[(y+1)%4][0],2)+pow(domain[x].nodes[y][1]-domain[x].nodes[(y+1)%4][1],2));
	}
}
