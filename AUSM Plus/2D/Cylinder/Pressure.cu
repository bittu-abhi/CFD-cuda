#include "hip/hip_runtime.h"
#include "ausmPlus.h"
#include <cmath>
#include <algorithm>

__global__ void pressureFlux(cell *domain, double *R, double *gammma)
{
	int y=threadIdx.x;
	int x=blockIdx.x;
	if(domain[x].flag==0 || domain[x].flag==4)
	{
		//Calculating the critical speed of sound for all the four sides/faces and the element itself
		double a_s[2];
		//Element
		a_s[0]=sqrt(2*(gammma[0]-1)/(gammma[0]+1)*(domain[x].stateVar[3]+(gammma[0]-1)*(domain[x].stateVar[3]\
			-0.5*(pow(domain[x].stateVar[1],2)+pow(domain[x].stateVar[2],2))/domain[x].stateVar[0])));
		//Side/face
		if(domain[x].flag!=4)
			a_s[1]=sqrt(2*(gammma[0]-1)/(gammma[0]+1)*(domain[(int)domain[x].face[y][0]].stateVar[3]+(gammma[0]-1)*(domain[(int)domain[x].face[y][0]].stateVar[3]\
			-0.5*(pow(domain[(int)domain[x].face[y][0]].stateVar[1],2)+pow(domain[(int)domain[x].face[y][0]].stateVar[2],2))/domain[(int)domain[x].face[y][0]].stateVar[0])));
		else
		{
			a_s[1]=a_s[0];
		}

		//speed for the boundary calculation
		a_s[0]=pow(a_s[0],2)/max(a_s[0],abs(sqrt(pow(domain[x].stateVar[1]/domain[x].stateVar[0],2)+pow(domain[x].stateVar[2]/domain[x].stateVar[0],2))));
		if(domain[x].flag!=4)
			a_s[1]=pow(a_s[0],2)/max(a_s[0],abs(sqrt(pow(domain[(int)domain[x].face[y][0]].stateVar[1]/domain[(int)domain[x].face[y][0]].stateVar[0],2)\
			+pow(domain[(int)domain[x].face[y][0]].stateVar[2]/domain[(int)domain[x].face[y][0]].stateVar[0],2))));
		else
		{
			a_s[1]=a_s[0];
		}

		int i1,i2;
		for (int i = 0; i < 4; ++i)
		{
			if(domain[x].nodes[i][2]==domain[x].face[y][0])
				i1=i;
			if(domain[x].nodes[(i+1)%4][2]==domain[x].face[(y+1)%4][0])
				i2=i+1;
		}
		//Speed of sound at facial interface
		double a_mid=min(a_s[0],a_s[1]);
		//Mach number of incoming and outgoing waves
		double machplus=(domain[x].stateVar[1]/domain[x].stateVar[0]*(domain[x].nodes[i1][0]-domain[x].nodes[i2][0])+\
		domain[x].stateVar[2]/domain[x].stateVar[0]*(domain[x].nodes[i2][1]-domain[x].nodes[i1][1]))/a_mid/sqrt(pow(domain[x].nodes[i1][0]-domain[x].nodes[i2][0],2)\
			+pow(domain[x].nodes[i2][1]-domain[x].nodes[i1][1],2));
		double machminus;
		if(domain[x].flag!=4)
		{
			machminus=-(domain[(int)domain[x].face[y][0]].stateVar[1]/domain[(int)domain[x].face[y][0]].stateVar[0]*(domain[x].nodes[i1][0]-domain[x].nodes[i2][0])+\
			domain[(int)domain[x].face[y][0]].stateVar[2]/domain[(int)domain[x].face[y][0]].stateVar[0]*(domain[x].nodes[i2][1]-domain[x].nodes[i1][1]))/a_mid/\
			sqrt(pow(domain[x].nodes[i1][0]-domain[x].nodes[i2][0],2)+pow(domain[x].nodes[i2][1]-domain[x].nodes[i1][1],2));
		}
			
		else
		{
			machminus=machplus;
		}

		//Pressure Fluxes
		double pressplus=domain[x].stateVar[3]+(gammma[0]-1)*(domain[x].stateVar[3]-0.5*(pow(domain[x].stateVar[1],2)+pow(domain[x].stateVar[2],2))/domain[x].stateVar[0]);
		double presminus;
		if(domain[x].flag!=4)
			presminus=domain[(int)domain[x].face[y][0]].stateVar[3]+(gammma[0]-1)*(domain[(int)domain[x].face[y][0]].stateVar[3]\
			-0.5*(pow(domain[(int)domain[x].face[y][0]].stateVar[1],2)+pow(domain[(int)domain[x].face[y][0]].stateVar[2],2))/domain[(int)domain[x].face[y][0]].stateVar[0]);
		else
		{
			presminus=domain[x].stateVar[3]+(gammma[0]-1)*(domain[x].stateVar[3]-0.5*(pow(domain[x].stateVar[1],2)+pow(domain[x].stateVar[2],2))/domain[x].stateVar[0]);
		}

		if(abs(machplus)>=1)
			pressplus*=0.5*(1+machplus/abs(machplus));
		else
			pressplus*=0.25*pow((machplus+1),2)*(2-machplus)+3/16*machplus*pow((pow(machplus,2)-1),2);
		if(abs(machminus)>=1)
			presminus*=0.5*(1-machminus/abs(machminus));
		else
			presminus*=0.25*pow((machminus-1),2)*(2+machminus)-3/16*machminus*pow((pow(machminus,2)-1),2);
		
		domain[x].presflux[y][0]=(pressplus+presminus)*(domain[x].nodes[i1][0]-domain[x].nodes[i2][0]);
		domain[x].presflux[y][1]=(pressplus+presminus)*(domain[x].nodes[i2][1]-domain[x].nodes[i1][1]);
	}
}