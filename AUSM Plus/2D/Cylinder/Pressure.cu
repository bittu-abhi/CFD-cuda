#include "hip/hip_runtime.h"
#include "ausmPlus.h"
#include <math.h>
#include <algorithm>
#include <stdio.h>

__global__ void pressureFlux(cell *domain, float *R, float *gammma)
{
	int y=threadIdx.x;
	int x=blockIdx.x;
	int ourFlag=(int)domain[x].flag;
	if(ourFlag==0 || ourFlag==4 || ourFlag==2)
	{


		//Calculating the critical speed of sound for all the four sides/faces and the element itself
		float a_s[2];

		//Element
		a_s[0]=sqrt(2*(gammma[0]-1)/(gammma[0]+1)*(domain[x].stateVar[3]+(gammma[0]-1)*(domain[x].stateVar[3]\
			-0.5*(powf(domain[x].stateVar[1],2)+powf(domain[x].stateVar[2],2))/domain[x].stateVar[0]))/domain[x].stateVar[0]);
		//Side/face
		a_s[1]=sqrt(2*(gammma[0]-1)/(gammma[0]+1)*(domain[x].temp_var[y][3]+(gammma[0]-1)*(domain[x].temp_var[y][3]\
			-0.5*(powf(domain[x].temp_var[y][1],2)+powf(domain[x].temp_var[y][2],2))/domain[x].temp_var[y][0]))/domain[x].temp_var[y][0]);
		
		//speed for the boundary calculation
		a_s[0]=powf(a_s[0],2)/max(a_s[0],abs(sqrt(powf(domain[x].stateVar[1]/domain[x].stateVar[0],2)+powf(domain[x].stateVar[2]/domain[x].stateVar[0],2))));
		a_s[1]=powf(a_s[1],2)/max(a_s[1],abs(sqrt(powf(domain[x].temp_var[y][1]/domain[x].temp_var[y][0],2)+powf(domain[x].temp_var[y][2]/domain[x].temp_var[y][0],2))));

		//Speed of sound at facial interface
		float a_mid=min(a_s[0],a_s[1]);

		//Mach number of incoming and outgoing waves
		float mach_one=(domain[x].stateVar[1]/domain[x].stateVar[0]*domain[x].norms[y][0]+domain[x].stateVar[2]/domain[x].stateVar[0]*domain[x].norms[y][1])/a_mid;
		float mach_two=(domain[x].temp_var[y][1]/domain[x].temp_var[y][0]*domain[x].norms[y][0]+domain[x].temp_var[y][2]/domain[x].temp_var[y][0]*domain[x].norms[y][1])/a_mid;
		
		//Pressure Fluxes
		float press_one=(*gammma-1)*(domain[x].stateVar[3]-0.5*(powf(domain[x].stateVar[1],2)+powf(domain[x].stateVar[2],2))/domain[x].stateVar[0]);
		float press_two=(*gammma-1)*(domain[x].temp_var[y][3]-0.5*(powf(domain[x].temp_var[y][1],2)+powf(domain[x].temp_var[y][2],2))/domain[x].temp_var[y][0]);

		float one,two;
		if(abs(mach_one)>=1)
			one=0.5*(1+mach_one/abs(mach_one));
		else
			one=0.25*powf((mach_one+1),2)*(2-mach_one)+3/16*mach_one*powf((powf(mach_one,2)-1),2);
	
		if(abs(mach_two)>=1)
			two=0.5*(1-mach_two/abs(mach_two));
		else
			two=0.25*powf((mach_two-1),2)*(2+mach_two)-3/16*mach_two*powf((powf(mach_two,2)-1),2);

		domain[x].presflux[y][0]=(press_one*one+press_two*two)*domain[x].norms[y][0]*sqrt(powf(domain[x].nodes[y][0]-domain[x].nodes[(y+1)%3][0],2)\
			+powf(domain[x].nodes[y][1]-domain[x].nodes[(y+1)%3][1],2));
		domain[x].presflux[y][1]=(press_one*one+press_two*two)*domain[x].norms[y][1]*sqrt(powf(domain[x].nodes[y][0]-domain[x].nodes[(y+1)%3][0],2)\
			+powf(domain[x].nodes[y][1]-domain[x].nodes[(y+1)%3][1],2));

		/*if(abs((1.0/3.0)*(domain[x].nodes[0][0]+domain[x].nodes[1][0]+domain[x].nodes[2][0])+0.524158619)<0.001 && abs((1.0/3.0)*(domain[x].nodes[0][1]+domain[x].nodes[1][1]+domain[x].nodes[2][1])-0.8526501336)<0.001)
		{
			printf("pressure  %5.14lf %5.14lf	 %d %d %d\n",domain[x].presflux[y][0],domain[x].presflux[y][1],domain[x].flag,x+1,y);
			//printf("mach(%5.14lf %5.14lf) pressure(%5.14lf %5.14lf) %d %d\n",one,two,press_one,press_two,x+1,y );

		}*/
	}
}
