#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cmath>
#include "ausmPlus.h"

using namespace std;

double gammma;
double mu;
double k;
double R;	

int main()
{
	gammma=1.4;
	mu=pow(1.789,-5);
	k=0.6065;
	R=186.9;
	
	double initial[4];
	//Rho
	initial[0]=1.225;
	//Rho*U
	initial[1]=200.000*initial[0];
	//Rho *V
	initial[2]=0.0000;
	//Rho*E, E is the internal energy including the kinetic energy(i.e. total intenal energy)
	initial[3]=(101325.000-0.5000*(pow(initial[1],2.0000)+pow(initial[2],2.0000))/initial[0])/(gammma-1.0000)+\
	0.5000*(pow(initial[1],2.0000)+pow(initial[2],2.0000))/initial[0];
	//Time steps and delta_t
	double timesteps=1000000;
	double deltat=0.00001;
	ausmplus(initial,timesteps,deltat);

	return 0;
}