#include "hip/hip_runtime.h"
#include "ausmPlus.h"
	
__global__ void diffusiveFlux(cell *domain,double *R, double *gammma, double *mu,double wall_temp,double *k)
{
	int x=blockIdx.x;
	int y=threadIdx.x;
	int note,faces=(int)domain[x].face[y][0];
	if(domain[x].flag==0 || domain[x].flag==4)
	{
		double x_cord[]={0,0},y_cord[]={0,0};
		if(domain[x].face[y][0]<1 || domain[x].face[y][0]>26000)
			note=y;
		int i1,i2;
		if(domain[x].flag!=4)
		{
			for (int i = 0; i < 4; ++i)
			{
				for (int j = 0; j < 4; ++j)
				{
					if(domain[x].nodes[i][0]==domain[faces].nodes[j][0] && domain[x].nodes[i][1]==domain[faces].nodes[j][1])
					{
						if(domain[x].nodes[(i+1)%4][0]==domain[faces].nodes[(j+1)%4][0] && domain[x].nodes[(i+1)%4][1]==domain[faces].nodes[(j+1)%4][1])
						{
							i1=i;
							i2=(i+1)%4;
						}
					}
				}
			}
		}
		else
		{
			for (int i = 0; i < 4; ++i)
			{
				if(domain[x].nodes[i][2]==domain[x].face[note][1] && domain[x].nodes[(i+1)%4][2]==domain[x].face[note][2])
				{
					i1=i;
					i2=(i+1)%4;
				}
			}
		}
		for (int i = 0; i < 4; ++i)
		{
			if(domain[x].flag!=4)
			{
				//x_cordinate of the elements
				x_cord[0]+=0.25*(domain[x].nodes[i][0]);
				x_cord[1]+=0.25*(domain[faces].nodes[i][0]);
				//Y coordinate of the elements
				y_cord[0]+=0.25*(domain[x].nodes[i][1]);
				y_cord[1]+=0.25*(domain[faces].nodes[i][1]);
			}
			else
			{
				//x_cordinate of the elements
				x_cord[0]+=0.25*(domain[x].nodes[i][0]);
				//Y coordinate of the elements
				y_cord[0]+=0.25*(domain[x].nodes[i][1]);
			}
		}
		if(domain[x].flag==4)
		{
			x_cord[1]=0.5*(domain[x].nodes[i1][0]+domain[x].nodes[i2][0]);
			y_cord[1]=0.5*(domain[x].nodes[i1][1]+domain[x].nodes[i2][1]);
		}
		double delu_delx,delv_delx,delu_dely,delv_dely;
		if(y!=note)
		{
			if(abs(x_cord[1]-x_cord[0])<=0.001)
			{
				delu_delx=0;
				delv_delx=0;
			}
			else
			{
				delu_delx=(domain[faces].stateVar[1]/domain[faces].stateVar[0]-domain[x].stateVar[1]/domain[x].stateVar[0])/(x_cord[1]-x_cord[0]);
				delv_delx=(domain[faces].stateVar[2]/domain[faces].stateVar[0]-domain[x].stateVar[2]/domain[x].stateVar[0])/(x_cord[1]-x_cord[0]);
			}
			if(abs(y_cord[1]-y_cord[0])<=0.001)
			{
				delu_dely=0;
				delv_dely=0;
			}
			else
			{
				delu_dely=(domain[faces].stateVar[1]/domain[faces].stateVar[0]-domain[x].stateVar[1]/domain[x].stateVar[0])/(y_cord[1]-y_cord[0]);
				delv_dely=(domain[faces].stateVar[2]/domain[faces].stateVar[0]-domain[x].stateVar[2]/domain[x].stateVar[0])/(y_cord[1]-y_cord[0]);
			}
		}
		else
		{
			if(abs(x_cord[1]-x_cord[0])<=0.001)
			{
				delu_delx=0;
				delv_delx=0;
			}
			else
			{
				delu_delx=(0-domain[x].stateVar[1]/domain[x].stateVar[0])/(x_cord[1]-x_cord[0]);
				delv_delx=(0-domain[x].stateVar[2]/domain[x].stateVar[0])/(x_cord[1]-x_cord[0]);
			}
			if(abs(y_cord[1]-y_cord[0])<=0.001)
			{
				delu_dely=0;
				delv_dely=0;
			}
			else
			{
				delu_dely=(0-domain[x].stateVar[1]/domain[x].stateVar[0])/(y_cord[1]-y_cord[0]);
				delv_dely=(0-domain[x].stateVar[2]/domain[x].stateVar[0])/(y_cord[1]-y_cord[0]);
			}
		}

		double tau_xx=2*mu[0]*(delu_delx-1/3*(delu_delx+delv_dely));
		double tau_yy=2*mu[0]*(delv_dely-1/3*(delu_delx+delv_dely));
		double tau_xy=mu[0]*(delu_dely+delv_delx);

		double temp[2];
		temp[0]=(gammma[0]-1)/R[0]*(domain[x].stateVar[3]-0.5*(pow(domain[x].stateVar[1],2)+pow(domain[x].stateVar[2],2))/domain[x].stateVar[0])/domain[x].stateVar[0];
		if(domain[x].flag!=4)
			temp[1]=(gammma[0]-1)/R[0]*(domain[faces].stateVar[3]-0.5*(pow(domain[faces].stateVar[1],2)\
				+pow(domain[faces].stateVar[2],2))/domain[faces].stateVar[0])/domain[faces].stateVar[0];
		else
		{
			temp[1]=wall_temp;
		}

		double delT_delx,delT_dely;
		if((x_cord[1]-x_cord[0])==0)
			delT_delx=0;
		else
			delT_delx=(temp[1]-temp[0])/(x_cord[1]-x_cord[0]);
		if((y_cord[1]-y_cord[0])==0)
			delT_dely=0;
		else
			delT_dely=(temp[1]-temp[0])/(y_cord[1]-y_cord[0]);

		double thetaX=domain[x].stateVar[1]/domain[x].stateVar[0]*tau_xx+domain[x].stateVar[2]/domain[x].stateVar[0]*tau_xy+k[0]*delT_delx;
		double thetaY=domain[x].stateVar[1]/domain[x].stateVar[0]*tau_xy+domain[x].stateVar[2]/domain[x].stateVar[0]*tau_yy+k[0]*delT_dely;

		if(domain[x].flag!=4)
		{
			domain[x].diffflux[y][0]=0;
			domain[x].diffflux[y][1]=(tau_xx*(domain[x].nodes[i1][0]-domain[x].nodes[i2][0])+tau_xy*(domain[x].nodes[i2][1]-domain[x].nodes[i1][1]));
			domain[x].diffflux[y][2]=(tau_xy*(domain[x].nodes[i1][0]-domain[x].nodes[i2][0])+tau_yy*(domain[x].nodes[i2][1]-domain[x].nodes[i1][1]));
			domain[x].diffflux[y][3]=(thetaX*(domain[x].nodes[i1][0]-domain[x].nodes[i2][0])+thetaY*(domain[x].nodes[i2][1]-domain[x].nodes[i1][1]));
		}
		else
		{
			domain[x].diffflux[y][0]=0;
			domain[x].diffflux[y][1]=(tau_xx*(domain[x].nodes[i1][0]-domain[x].nodes[i2][0])+tau_xy*(domain[x].nodes[i2][1]-domain[x].nodes[i1][1]));
			domain[x].diffflux[y][2]=(tau_xy*(domain[x].nodes[i1][0]-domain[x].nodes[i2][0])+tau_yy*(domain[x].nodes[i2][1]-domain[x].nodes[i1][1]));
			domain[x].diffflux[y][3]=(thetaX*(domain[x].nodes[i1][0]-domain[x].nodes[i2][0])+thetaY*(domain[x].nodes[i2][1]-domain[x].nodes[i1][1]));
		}
	}
}
