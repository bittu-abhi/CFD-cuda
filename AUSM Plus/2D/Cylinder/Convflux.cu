#include "hip/hip_runtime.h"
#include "ausmPlus.h"
#include <math.h>


__global__ void convectiveflux(cell *domain, double *R, double *gammma)
{
	int y=threadIdx.x;
	int x=blockIdx.x;
	int ourFlag=(int)domain[x].flag;
	if(ourFlag==0 || ourFlag==4)
	{
		//Calculating the critical speed of sound for all the four sides/faces and the element itself
		double a_s[2];

		//Element
		a_s[0]=sqrt(2*(gammma[0]-1)/(gammma[0]+1)*(domain[x].stateVar[3]+(gammma[0]-1)*(domain[x].stateVar[3]\
			-0.5*(pow(domain[x].stateVar[1],2)+pow(domain[x].stateVar[2],2))/domain[x].stateVar[0]))/domain[x].stateVar[0]);
		//Side/face
		a_s[1]=sqrt(2*(gammma[0]-1)/(gammma[0]+1)*(domain[x].temp_var[y][3]+(gammma[0]-1)*(domain[x].temp_var[y][3]\
			-0.5*(pow(domain[x].temp_var[y][1],2)+pow(domain[x].temp_var[y][2],2))/domain[x].temp_var[y][0]))/domain[x].temp_var[y][0]);

		//speed for the boundary calculation
		a_s[0]=pow(a_s[0],2)/max(a_s[0],abs(sqrt(pow(domain[x].stateVar[1]/domain[x].stateVar[0],2)+pow(domain[x].stateVar[2]/domain[x].stateVar[0],2))));
		a_s[1]=pow(a_s[1],2)/max(a_s[1],abs(sqrt(pow(domain[x].temp_var[y][1]/domain[x].temp_var[y][0],2)+pow(domain[x].temp_var[y][2]/domain[x].temp_var[y][0],2))));

		//Speed of sound at facial interface
		double a_mid=min(a_s[0],a_s[1]);

		//Pressure
		double press=domain[x].stateVar[3]+(*gammma-1)*(domain[x].stateVar[3]-0.5*(pow(domain[x].stateVar[1],2)+pow(domain[x].stateVar[2],2))/domain[x].stateVar[0]);

		//Machnumber of contravarient velocity(V=u*nx+v*ny)
		double machplus=(domain[x].stateVar[1]/domain[x].stateVar[0]*domain[x].norms[y][0]+domain[x].stateVar[2]/domain[x].stateVar[0]*domain[x].norms[y][1])/a_mid;
		double machminus=(domain[x].temp_var[y][1]/domain[x].temp_var[y][0]*domain[x].norms[y][0]+domain[x].temp_var[y][2]/domain[x].temp_var[y][0]*domain[x].norms[y][1])/a_mid;

		double split_mach_plus,split_mach_minus;

		if(abs(machplus)>=1)
			split_mach_plus=0.5*(machplus+abs(machplus));
		else
			split_mach_plus=0.5*pow(machplus+1.0,2.0)+1/8*pow(pow(machplus,2.0)-1.0,2.0);
		if(abs(machminus)>=1)
			split_mach_minus=0.5*(machminus-abs(machminus));
		else
			split_mach_minus=-0.5*pow(machminus-1.0,2.0)-1/8*pow(pow(machminus,2.0)-1.0,2.0);

		double split_mach=split_mach_plus+split_mach_minus;

		for (int i = 0; i < 4; ++i)
		{
			domain[x].convflux[y][i]=a_mid*(0.5*(split_mach+abs(split_mach))*domain[x].stateVar[i]+0.5*(split_mach-abs(split_mach))\
				*domain[x].temp_var[y][i])*sqrt(pow(domain[x].nodes[y][0]-domain[x].nodes[(y+1)%4][0],2)+pow(domain[x].nodes[y][1]-domain[x].nodes[(y+1)%4][1],2));
		}
		domain[x].convflux[y][3]+=a_mid*(0.5*(split_mach+abs(split_mach))*press+0.5*(split_mach-abs(split_mach))*press)\
		*sqrt(pow(domain[x].nodes[y][0]-domain[x].nodes[(y+1)%4][0],2)+pow(domain[x].nodes[y][1]-domain[x].nodes[(y+1)%4][1],2));
		/*if((ourFlag==4 && domain[x].face[y]<1) || (ourFlag==4 && domain[x].face[y]>24542))
		{
			domain[x].convflux[y][0]=0;
			domain[x].convflux[y][3]=0;
			domain[x].convflux[y][1]=0;
			domain[x].convflux[y][2]=0;
		}*/
	}
}
