#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include "sbli.h"

void WENO(point *pt,int points,int flagXY)
{
	double cons[3] = {1/10,3/5,3/10};
	double beta[4][3]={{0},{0},{0}};
	int x = blockIdx.x;
	int y = blockIdx.y;
	int state = threadIdx.x;

	beta[state][0] = 13/12*pow((pt[(flagXY==0):(x-2+y*points)?(x+(y-2)*points)].stateVar[state]-2*pt[(flagXY==0):(x-1+y*points)?(x+(y-1)*points)].stateVar[state]+pt[(flagXY==0):(x+y*points)?(x+(y)*points)].stateVar[state]),2)+1/4*pow((pt[(flagXY==0):(x-2+y*points)?(x+(y-2)*points)].stateVar[state]-4*pt[(flagXY==0):(x-1+y*points)?(x+(y-1)*points)].stateVar[state]+3*pt[(flagXY==0):(x+y*points)?(x+(y)*points)].stateVar[state]),2);

	beta[state][1] = 13/12*pow((pt[(flagXY==0):(x-1+y*points)?(x+(y-1)*points)].stateVar[state]-2*pt[(flagXY==0):(x+y*points)?(x+(y)*points)].stateVar[state]+pt[(flagXY==0):(x+1+y*points)?(x+(y+1)*points)].stateVar[state]),2)+1/4*pow((pt[(flagXY==0):(x-1+y*points)?(x+(y-1)*points)].stateVar[state]-pt[(flagXY==0):(x+1+y*points)?(x+(y+1)*points)].stateVar[state]),2);

	beta[state][2] = 13/12*pow((pt[(flagXY==0):(x+y*points)?(x+(y)*points)].stateVar[state]-2*pt[(flagXY==0):(x+1+y*points)?(x+(y+1)*points)].stateVar[state]+pt[(flagXY==0):(x+2+y*points)?(x+(y+2)*points)].stateVar[state]),2)+1/4*pow((3*pt[(flagXY==0):(x+y*points)?(x+(y)*points)].stateVar[state]-4*pt[(flagXY==0):(x+1+y*points)?(x+(y+1)*points)].stateVar[state]+pt[(flagXY==0):(x+2+y*points)?(x+(y+2)*points)].stateVar[state]),2);

	double wtilda[4][3]={{0},{0},{0}};

	wtilda[state][0] = (1/10)/pow((beta[state][0]+0.000001),2);
	wtilda[state][1] = (3/5)/pow((beta[state][1]+0.000001),2);
	wtilda[state][2] = (3/10)/pow((beta[state][2]+0.000001),2);

	double deno = wtilda[state][0]+wtilda[state][1]+wtilda[state][2];

	double weight[4][3] = {{wtilda[state][0]/deno},{wtilda[state][1]/deno},{wtilda[state][2]/deno}};

	double recon_poly[4][3] =  {{0},{0},{0}};

	recon_poly[state][0] = 1/3*(pt[(flagXY==0):(x-2+y*points)?(x+(y-2)*points)].stateVar[state])-7/6*(pt[(flagXY==0):(x-1+y*points)?(x+(y-1)*points)].stateVar[state])+11/6*(pt[(flagXY==0):(x+y*points)?(x+(y)*points)].stateVar[state]);

	recon_poly[state][1] = -1/6*(pt[(flagXY==0):(x-1+y*points)?(x+(y-1)*points)].stateVar[state])+5/6*(pt[(flagXY==0):(x+y*points)?(x+(y)*points)].stateVar[state])+1/3*(pt[(flagXY==0):(x+1+y*points)?(x+(y+1)*points)].stateVar[state]);

	recon_poly[state][2] = 1/3*(pt[(flagXY==0):(x+y*points)?(x+(y)*points)].stateVar[state])+5/6*(pt[(flagXY==0):(x+1+y*points)?(x+(y+1)*points)].stateVar[state])-1/6*(pt[(flagXY==0):(x+2+y*points)?(x+(y+2)*points)].stateVar[state]);

	pt[x+y*points].interface[state][flagXY] = weight[state][0]*recon_poly[state][0] + weight[state][1]*recon_poly[state][1] + weight[state][2]*recon_poly[state][2]; //It can be returned. Same for i-1/2
}

