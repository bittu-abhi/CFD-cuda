#include "hip/hip_runtime.h"
#include "lid.h"
#include <stdio.h>
#ifndef CUR
#define CUR curr[y+x*(points)]
#endif

using namespace std;

__global__ void timeW(point *curr, double deltat, double re, int points)
{
	int x = blockIdx.x;
	int y = threadIdx.x;
	/*
	//This part of the code requires some back of the copy formulations hence I am currently using the forward time scheme

	//LDDRK scheme is used
	double k[4];
	//coefficiens of LDDRK scheme
	double c[]={1,0.5,0.162997,0.0407574};
	double beta[]={0,0,0,c[1]};
	beta[2] = c[2]/beta[3];
	beta[1] = c[3]/(beta[3]*beta[2]);

	k[0] = deltat*curr[y+x*sizeof(point)].w;
	k[1] = deltat*

	*/
	CUR.w=CUR.w-deltat*((CUR.delW[0]*CUR.delpsi[1]-CUR.delW[1]*CUR.delpsi[0])-(1/re)*(CUR.delW2[0]+CUR.delW2[1]));
	if(x==points-1 && y == points-1)
		printf("w=%5.14lf psi=%5.14lf delW2X=%5.14lf delW2Y=%5.14lf\n",CUR.w,CUR.psi,CUR.delW2[0],CUR.delW2[1]);
}