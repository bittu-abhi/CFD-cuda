#include "hip/hip_runtime.h"
#include "ausmPlus.h"
#include <hip/hip_runtime_api.h>
#include <math.h>
#include <stdio.h>

__global__ void set_nodes(float *node, cell *domain, float *boundary,float *initial,float *gammma)
{
	int x=blockIdx.x;
	int y=threadIdx.x;
	int temp=3*((int)(domain[x].nodes[y][2])-1);
	domain[x].nodes[y][1]=node[temp+1];
	domain[x].nodes[y][0]=node[temp];
	
	//Inflow
	if(domain[x].nodes[y][2]>500 && domain[x].nodes[y][2]<601 || domain[x].nodes[y][2]==1)
	{
		if((domain[x].nodes[(y+1)%3][2]>500 && domain[x].nodes[(y+1)%3][2]<601 || domain[x].nodes[(y+1)%3][2]==1)||(domain[x].nodes[(y-1+3)%3][2]>500 && domain[x].nodes[(y-1+3)%3][2]<601 || domain[x].nodes[(y+1)%3][2]==1))
		{
			domain[x].flag=1;
			domain[x].stateVar[0]=initial[0];
			domain[x].stateVar[1]=initial[1];
			domain[x].stateVar[2]=initial[2];
			domain[x].stateVar[3]=(101325.000-0.5000*(powf(initial[1],2.0000)+powf(initial[2],2.0000))/initial[0])/(*gammma-1.0000)+\
		0.5000*(powf(initial[1],2.0000)+powf(initial[2],2.0000))/initial[0];	
		}
	}
	
	//Outflow
	if(domain[x].nodes[y][2]>200 && domain[x].nodes[y][2]<302)
	{
		if((domain[x].nodes[(y+1)%3][2]>200 && domain[x].nodes[(y+1)%3][2]<302)|| (domain[x].nodes[(y-1+3)%3][2]>200 && domain[x].nodes[(y-1+3)%3][2]<302))
		{
			domain[x].flag=2;
		}
	}

	//Assumed Farfield
	if((domain[x].nodes[y][2]>301 && domain[x].nodes[y][2]<501) || (domain[x].nodes[y][2]>1 && domain[x].nodes[y][2]<201))
	{
		if(((domain[x].nodes[(y+1)%3][2]>301 && domain[x].nodes[(y+1)%3][2]<501) || (domain[x].nodes[(y+1)%3][2]>1 && domain[x].nodes[(y+1)%3][2]<201))||((domain[x].nodes[(y-1+3)%3][2]>301 && domain[x].nodes[(y-1+3)%3][2]<501) || (domain[x].nodes[(y-1+3)%3][2]>1 && domain[x].nodes[(y-1+3)%3][2]<201)))
		{
			domain[x].flag=2;
			/*domain[x].stateVar[0]=initial[0];
			domain[x].stateVar[1]=initial[1];
			domain[x].stateVar[2]=initial[2];
			domain[x].stateVar[3]=(101325.000-0.5000*(powf(initial[1],2.0000)+powf(initial[2],2.0000))/initial[0])/(*gammma-1.0000)+\
			0.5000*(powf(initial[1],2.0000)+powf(initial[2],2.0000))/initial[0];*/
		}	
	}

	//Cylinder Wall
	if(domain[x].nodes[y][2]>600 && domain[x].nodes[y][2]<2201)
	{
		if((domain[x].nodes[(y+1)%3][2]>600 && domain[x].nodes[(y+1)%3][2]<2201)||(domain[x].nodes[(y-1+3)%3][2]>600 && domain[x].nodes[(y-1+3)%3][2]<2201))
		{
			domain[x].flag=4;
		}
	}
	
	//Fluid Region
	if(domain[x].flag!=1 && domain[x].flag!=2 && domain[x].flag!=3 && domain[x].flag!=4)
		domain[x].flag=0;

	if ((domain[x].nodes[y][2]==301 && domain[x].nodes[(y+1)%3][2]==302) || (domain[x].nodes[y][2]==301 && domain[x].nodes[(y+3-1)%3][2]==302))
		domain[x].flag=2;

	if ((domain[x].nodes[y][2]==200 && domain[x].nodes[(y+1)%3][2]==201) || 
		(domain[x].nodes[y][2]==200 && domain[x].nodes[(y+3-1)%3][2]==201))
		domain[x].flag=2;
}

__global__ void set_neighbour(cell *domain)
{
	int x=blockIdx.x;
	int y=threadIdx.x;
	int i,flag1=0,flag2=0;
	for (i = 0; i < 50266; i+=1)
	{
		for (int j = 0; j <3; j+=1)
		{
			if(domain[i].nodes[j][0]==domain[x].nodes[y][0] && i!=x && domain[i].nodes[j][1]==domain[x].nodes[y][1])
				flag1=1;
			if( domain[i].nodes[j][0]==domain[x].nodes[(y+1)%3][0] && i!=x && domain[i].nodes[j][1]==domain[x].nodes[(y+1)%3][1])
				flag2=1;
		}
		if(flag1==1 && flag2==1)
		{	
			domain[x].face[y]=i+1;
			break;
		}
		flag1=0;
		flag2=0;
	}
}

__global__ void calculate_norm(cell *domain)
{
	int x=blockIdx.x;
	int y=threadIdx.x;

	//Now to determine if the normal is pointing outward, and if not, then change accordingly
	float cen_cord[2];
	cen_cord[0]=(1.0/3.0)*(domain[x].nodes[0][0]+domain[x].nodes[1][0]+domain[x].nodes[2][0]);
	cen_cord[1]=(1.0/3.0)*(domain[x].nodes[0][1]+domain[x].nodes[1][1]+domain[x].nodes[2][1]);

	//construct the face
	float m,c;
	m=(domain[x].nodes[(y+1)%3][1]-domain[x].nodes[y][1])/(domain[x].nodes[(y+1)%3][0]-domain[x].nodes[y][0]);

	c=domain[x].nodes[y][1]-m*domain[x].nodes[y][0];

	//A perpendicular line passing through the centre of the element
	if(m!=0.0000 && !isinf(m))
	{
		float req_m=-1/m;
		float req_c=cen_cord[1]-req_m*cen_cord[0];

		//Intersection of this line with the face would give a point on the face. Now using this point as (x1,y2), we would
		//always get a vector pointing outward from the face,regardless of the way the nodes are number(clockwise or anticlockwise)
		float req_x=(c-req_c)/(req_m-m);
		float req_y=m*req_x+c;
		
		domain[x].norms[y][0]=(req_x-cen_cord[0]);
		domain[x].norms[y][1]=(req_y-cen_cord[1]);

		float dino=sqrt(powf((req_x-cen_cord[0]),2)+powf((req_y-cen_cord[1]),2));
		
		domain[x].norms[y][0]/=dino;
		domain[x].norms[y][1]/=dino;		

	}
	else if(m==0.0000)
	{
		domain[x].norms[y][0]=0;
		if(domain[x].nodes[y][1]<cen_cord[1])
			domain[x].norms[y][1]=-1.000;
		else 
			domain[x].norms[y][1]=1.000;
	}
	else
	{
		domain[x].norms[y][1]=0;
		if(domain[x].nodes[y][0]<cen_cord[0])
			domain[x].norms[y][0]=-1.000;
		else
			domain[x].norms[y][0]=1.0000;
	}
}

__global__ void read_values(cell *domain)
{
	int x=blockIdx.x;
	int y=threadIdx.x;
	int faces=(int)domain[x].face[y]-1;
	int note=-10;

	if(faces<0 || faces >50266)
	{
		note=y;
	}
	if(y!=note)
	{
		for (int i = 0; i < 4; ++i)
		{
			domain[x].temp_var[y][i]=domain[faces].stateVar[i];
		}
		if(note!=y && domain[x].flag==2)
		{
			domain[x].temp_var[y][0]=1.225;
			domain[x].temp_var[y][1]=domain[(int)domain[x].face[y]-1].stateVar[1];
			domain[x].temp_var[y][2]=domain[(int)domain[x].face[y]-1].stateVar[2];
			domain[x].temp_var[y][3]=domain[(int)domain[x].face[y]-1].stateVar[3];
		}
	}
	else
	{
		if(domain[x].flag==4)
		{
			domain[x].temp_var[note][0]=1.225;
			domain[x].temp_var[note][1]=-1.0000*domain[x].stateVar[1];
			domain[x].temp_var[note][2]=-1.0000*domain[x].stateVar[2];
			domain[x].temp_var[note][3]=domain[x].stateVar[3];
		}
		if(domain[x].flag==2)
		{
			domain[x].temp_var[y][0]=domain[x].stateVar[0];
			domain[x].temp_var[y][1]=domain[x].stateVar[1];
			domain[x].temp_var[y][2]=domain[x].stateVar[2];
			domain[x].temp_var[y][3]=domain[x].stateVar[3];
		}
	}
}
