#include "hip/hip_runtime.h"
#include "ausmPlus.h"
#include <math.h>
#include <stdio.h>

__global__ void convectiveflux(cell *domain, double *R, double *gammma)
{
	int y=threadIdx.x;
	int x=blockIdx.x;
	if(domain[x].flag==0 || domain[x].flag==4)
	{
		//Calculating the critical speed of sound for all the four sides/faces and the element itself
		double a_s[2];
		//Element
		a_s[0]=sqrt(2*(gammma[0]-1)/(gammma[0]+1)*(domain[x].stateVar[3]+(gammma[0]-1)*(domain[x].stateVar[3]\
			-0.5*(pow(domain[x].stateVar[1],2)+pow(domain[x].stateVar[2],2))/domain[x].stateVar[0])));
		//Side/face
		if(domain[x].flag!=4)
			a_s[1]=sqrt(2*(gammma[0]-1)/(gammma[0]+1)*(domain[(int)domain[x].face[y][0]].stateVar[3]+(gammma[0]-1)*(domain[(int)domain[x].face[y][0]].stateVar[3]\
			-0.5*(pow(domain[(int)domain[x].face[y][0]].stateVar[1],2)+pow(domain[(int)domain[x].face[y][0]].stateVar[2],2))/domain[(int)domain[x].face[y][0]].stateVar[0])));
		else
		{
			a_s[1]=a_s[0];
		}

		//speed for the boundary calculation
		a_s[0]=pow(a_s[0],2)/max(a_s[0],abs(sqrt(pow(domain[x].stateVar[1]/domain[x].stateVar[0],2)+pow(domain[x].stateVar[2]/domain[x].stateVar[0],2))));
		if(domain[x].flag!=4)
			a_s[1]=pow(a_s[0],2)/max(a_s[0],abs(sqrt(pow(domain[(int)domain[x].face[y][0]].stateVar[1]/domain[(int)domain[x].face[y][0]].stateVar[0],2)\
			+pow(domain[(int)domain[x].face[y][0]].stateVar[2]/domain[(int)domain[x].face[y][0]].stateVar[0],2))));
		else
			a_s[1]=a_s[0];
		
		//Speed of sound at facial interface
		double a_mid=min(a_s[0],a_s[1]);

		//Pressure Fluxes
		double pressplus=domain[x].stateVar[3]+(*gammma-1)*(domain[x].stateVar[3]-0.5*(pow(domain[x].stateVar[1],2)+pow(domain[x].stateVar[2],2))/domain[x].stateVar[0]);
		double presminus;
		if(domain[x].flag!=4)
			presminus=domain[(int)domain[x].face[y][0]].stateVar[3]+(*gammma-1)*(domain[(int)domain[x].face[y][0]].stateVar[3]\
			-0.5*(pow(domain[(int)domain[x].face[y][0]].stateVar[1],2)+pow(domain[(int)domain[x].face[y][0]].stateVar[2],2))/domain[(int)domain[x].face[y][0]].stateVar[0]);
		else
			presminus=domain[x].stateVar[3]+(gammma[0]-1)*(domain[x].stateVar[3]-0.5*(pow(domain[x].stateVar[1],2)+pow(domain[x].stateVar[2],2))/domain[x].stateVar[0]);

		int i1,i2;
		for (int i = 0; i < 4; ++i)
		{
			if(domain[x].nodes[i][2]==domain[x].face[y][0])
				i1=i;
			if(domain[x].nodes[(i+1)%4][2]==domain[x].face[(y+1)%4][0])
				i2=i+1;
		}
		//Machnumber of contravarient velocity
		double machplus=(domain[x].stateVar[1]/domain[x].stateVar[0]*(domain[x].nodes[i1][0]-domain[x].nodes[i2][0])+\
		domain[x].stateVar[2]/domain[x].stateVar[0]*(domain[x].nodes[i2][1]-domain[x].nodes[i1][1]))/a_mid/sqrt(pow(domain[x].nodes[i1][0]-domain[x].nodes[i2][0],2)\
			+pow(domain[x].nodes[i2][1]-domain[x].nodes[i1][1],2));
		double machminus;
		if(domain[x].flag!=4)
		{
			machminus=-(domain[(int)domain[x].face[y][0]].stateVar[1]/domain[(int)domain[x].face[y][0]].stateVar[0]*(domain[x].nodes[i1][0]-domain[x].nodes[i2][0])+\
			domain[(int)domain[x].face[y][0]].stateVar[2]/domain[(int)domain[x].face[y][0]].stateVar[0]*(domain[x].nodes[i2][1]-domain[x].nodes[i1][1]))/a_mid/\
			sqrt(pow(domain[x].nodes[i1][0]-domain[x].nodes[i2][0],2)+pow(domain[x].nodes[i2][1]-domain[x].nodes[i1][1],2));
		}
			
		else
		{
			machminus=machplus;
		}
		double split_mach_plus,split_mach_minus;

		if(abs(machplus)>=1)
			split_mach_plus=0.5*(machplus+abs(machplus));
		else
			split_mach_plus=0.5*pow(machplus+1.0,2.0)+1/8*pow(pow(machplus,2.0)-1.0,2.0);
		if(abs(machminus)>=1)
			split_mach_minus=0.5*(machminus-abs(machminus));
		else
			split_mach_minus=-0.5*pow(machminus-1.0,2.0)-1/8*pow(pow(machminus,2.0)-1.0,2.0);

		if(domain[x].flag!=4)
		{
			for (int i = 0; i < 4; ++i)
			{
				domain[x].convflux[y][i]=a_mid*(0.5*(split_mach_plus+abs(split_mach_plus))*domain[x].stateVar[i]+0.5*(split_mach_minus-abs(split_mach_minus))\
					*domain[(int)domain[x].face[y][0]].stateVar[i]);
			}
			domain[x].convflux[y][3]+=a_mid*(0.5*(split_mach_plus+abs(split_mach_plus))*pressplus+0.5*(split_mach_minus-abs(split_mach_minus))*presminus);
		}
		else
		{
			for (int i = 0; i < 4; ++i)
			{
				domain[x].convflux[y][i]=a_mid*(0.5*(split_mach_plus+abs(split_mach_plus))*domain[x].stateVar[i]+0.5*(split_mach_minus-abs(split_mach_minus))\
					*domain[(int)domain[x].face[y][0]].stateVar[i]);
				if(i==1 || i==2)
					domain[x].convflux[y][i]=0;
				
			}
			domain[x].convflux[y][3]+=a_mid*(0.5*(split_mach_plus+abs(split_mach_plus))*pressplus+0.5*(split_mach_minus-abs(split_mach_minus))*presminus);
		}
	}
}