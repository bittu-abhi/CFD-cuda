#include "hip/hip_runtime.h"
#include "ausmPlus.h"
#include <math.h>
#include <stdio.h>

__global__ void convectiveflux(cell *domain, double *R, double *gammma)
{
	int y=threadIdx.x;
	int x=blockIdx.x;
	int faces=(int)domain[x].face[y];
	int ourFlag=(int)domain[x].flag;
	int note=-1;
	if(ourFlag==0 || ourFlag==4)
	{
		//Calculating the critical speed of sound for all the four sides/faces and the element itself
		double a_s[2];

		if(domain[x].face[y]<1)
		{
			note=y;
		}

		//Element
		a_s[0]=sqrt(2*(gammma[0]-1)/(gammma[0]+1)*(domain[x].stateVar[3]+(gammma[0]-1)*(domain[x].stateVar[3]\
			-0.5*(pow(domain[x].stateVar[1],2)+pow(domain[x].stateVar[2],2))/domain[x].stateVar[0])));
		//Side/face
		if((int)ourFlag!=4 || ((int)ourFlag==4 && y!=note))
			a_s[1]=sqrt(2*(gammma[0]-1)/(gammma[0]+1)*(domain[faces].stateVar[3]+(gammma[0]-1)*(domain[faces].stateVar[3]\
			-0.5*(pow(domain[faces].stateVar[1],2)+pow(domain[faces].stateVar[2],2))/domain[faces].stateVar[0])));
		else
		{
			a_s[1]=a_s[0];
		}

		//speed for the boundary calculation
		a_s[0]=pow(a_s[0],2)/max(a_s[0],abs(sqrt(pow(domain[x].stateVar[1]/domain[x].stateVar[0],2)+pow(domain[x].stateVar[2]/domain[x].stateVar[0],2))));
		if(ourFlag!=4 || (ourFlag==4 && y!=note))
			a_s[1]=pow(a_s[1],2)/max(a_s[1],abs(sqrt(pow(domain[faces].stateVar[1]/domain[faces].stateVar[0],2)\
			+pow(domain[faces].stateVar[2]/domain[faces].stateVar[0],2))));
		else
			a_s[1]=a_s[0];

		//Speed of sound at facial interface
		double a_mid=min(a_s[0],a_s[1]);

		//Pressure
		double press=domain[x].stateVar[3]+(*gammma-1)*(domain[x].stateVar[3]-0.5*(pow(domain[x].stateVar[1],2)+pow(domain[x].stateVar[2],2))/domain[x].stateVar[0]);

		//Machnumber of contravarient velocity(V=u*nx+v*ny)
		double machplus=(domain[x].stateVar[1]/domain[x].stateVar[0]*domain[x].norms[y][0]+domain[x].stateVar[2]/domain[x].stateVar[0]*domain[x].norms[y][1])/a_mid;
		double machminus;
		if(ourFlag!=4 || (ourFlag==4 && y!=note))
		{
			machminus=(domain[faces].stateVar[1]/domain[faces].stateVar[0]*domain[x].norms[y][0]+domain[faces].stateVar[2]/domain[faces].stateVar[0]*domain[x].norms[y][1])/a_mid;
		}
		else
		{
			machminus=-machplus;
		}

		double split_mach_plus,split_mach_minus;

		if(abs(machplus)>=1)
			split_mach_plus=0.5*(machplus+abs(machplus));
		else
			split_mach_plus=0.5*pow(machplus+1.0,2.0)+1/8*pow(pow(machplus,2.0)-1.0,2.0);
		if(abs(machminus)>=1)
			split_mach_minus=0.5*(machminus-abs(machminus));
		else
			split_mach_minus=-0.5*pow(machminus-1.0,2.0)-1/8*pow(pow(machminus,2.0)-1.0,2.0);

		if(ourFlag!=4 || (ourFlag==4 && y!=note))
		{
			for (int i = 0; i < 4; ++i)
			{
				domain[x].convflux[y][i]=a_mid*(0.5*(split_mach_plus+abs(split_mach_plus))*domain[x].stateVar[i]+0.5*(split_mach_minus-abs(split_mach_minus))\
					*domain[faces].stateVar[i])*sqrt(pow(domain[x].nodes[y][0]-domain[x].nodes[(y+1)%4][0],2)+pow(domain[x].nodes[y][1]-domain[x].nodes[(y+1)%4][1],2));
			}
			domain[x].convflux[y][3]+=a_mid*(0.5*(split_mach_plus+abs(split_mach_plus))*press+0.5*(split_mach_minus-abs(split_mach_minus))*press)\
			*sqrt(pow(domain[x].nodes[y][0]-domain[x].nodes[(y+1)%4][0],2)+pow(domain[x].nodes[y][1]-domain[x].nodes[(y+1)%4][1],2));
		}
		else
		{
			for (int i = 0; i < 4; ++i)
			{
				if(i==1 || i==2)
					domain[x].convflux[y][i]=a_mid*(0.5*(split_mach_plus+abs(split_mach_plus))*domain[x].stateVar[i]-0.5*(split_mach_minus-abs(split_mach_minus))\
					*domain[x].stateVar[i])*sqrt(pow(domain[x].nodes[y][0]-domain[x].nodes[(y+1)%4][0],2)+pow(domain[x].nodes[y][1]-domain[x].nodes[(y+1)%4][1],2));
				else
					domain[x].convflux[y][i]=a_mid*(0.5*(split_mach_plus+abs(split_mach_plus))*domain[x].stateVar[i]+0.5*(split_mach_minus-abs(split_mach_minus))\
					*domain[x].stateVar[faces])*sqrt(pow(domain[x].nodes[y][0]-domain[x].nodes[(y+1)%4][0],2)+pow(domain[x].nodes[y][1]-domain[x].nodes[(y+1)%4][1],2));
			}
			domain[x].convflux[y][3]+=a_mid*(0.5*(split_mach_plus+abs(split_mach_plus))*press+0.5*(split_mach_minus-abs(split_mach_minus))*press)\
			*sqrt(pow(domain[x].nodes[y][0]-domain[x].nodes[(y+1)%4][0],2)+pow(domain[x].nodes[y][1]-domain[x].nodes[(y+1)%4][1],2));
		}
		//if(domain[x].flag==4)
		//	printf("%lf %lf %d %d\n",domain[x].convflux[y][1],domain[x].convflux[y][2],x,y);
	}
}
