#include "hip/hip_runtime.h"
#include "ausmPlus.h"
#include <math.h>
#include <algorithm>

__global__ void pressureFlux(cell *domain, double *R, double *gammma)
{
	int y=threadIdx.x;
	int x=blockIdx.x;
	int faces=(int)domain[x].face[y];
	int note=-1;
	int ourFlag=(int)domain[x].flag;
	if(ourFlag==0 || ourFlag==4)
	{

		//Calculating the critical speed of sound for all the four sides/faces and the element itself
		double a_s[2];
		if(domain[x].face[y]<1)
		{
			note=y;
		}
		
		//Element
		a_s[0]=sqrt(2*(gammma[0]-1)/(gammma[0]+1)*(domain[x].stateVar[3]+(gammma[0]-1)*(domain[x].stateVar[3]\
			-0.5*(pow(domain[x].stateVar[1],2)+pow(domain[x].stateVar[2],2))/domain[x].stateVar[0])));
		//Side/face
		if(ourFlag!=4 || (ourFlag==4 && y!=note))
			a_s[1]=sqrt(2*(gammma[0]-1)/(gammma[0]+1)*(domain[faces].stateVar[3]+(gammma[0]-1)*(domain[faces].stateVar[3]\
			-0.5*(pow(domain[faces].stateVar[1],2)+pow(domain[faces].stateVar[2],2))/domain[faces].stateVar[0])));
		else
		{
			a_s[1]=a_s[0];
		}
		
		//speed for the boundary calculation
		a_s[0]=pow(a_s[0],2)/max(a_s[0],abs(sqrt(pow(domain[x].stateVar[1]/domain[x].stateVar[0],2)+pow(domain[x].stateVar[2]/domain[x].stateVar[0],2))));
		if(ourFlag!=4 || (ourFlag==4 && y!=note))
			a_s[1]=pow(a_s[1],2)/max(a_s[1],abs(sqrt(pow(domain[faces].stateVar[1]/domain[faces].stateVar[0],2)+pow(domain[faces].stateVar[2]/domain[faces].stateVar[0],2))));
		else
		{
			a_s[1]=a_s[0];
		}

		//Speed of sound at facial interface
		double a_mid=min(a_s[0],a_s[1]);
		//Mach number of incoming and outgoing waves
		double machplus=(domain[x].stateVar[1]/domain[x].stateVar[0]*domain[x].norms[y][0]+domain[x].stateVar[2]/domain[x].stateVar[0]*domain[x].norms[y][1])/a_mid;
		double machminus;
		if(ourFlag!=4 || (ourFlag==4 && y!=note))
		{
			machminus=(domain[faces].stateVar[1]/domain[faces].stateVar[0]*domain[x].norms[y][0]+domain[faces].stateVar[2]/domain[faces].stateVar[0]*domain[x].norms[y][1])/a_mid;
		}
		else
		{
			machminus=-machplus;
		}
		
		//Pressure Fluxes
		double pressplus=(gammma[0]-1)*(domain[x].stateVar[3]-0.5*(pow(domain[x].stateVar[1],2)+pow(domain[x].stateVar[2],2))/domain[x].stateVar[0]);
		double presminus;
		if(ourFlag!=4 || (ourFlag==4 && y!=note))
			presminus=(gammma[0]-1)*(domain[faces].stateVar[3]-0.5*(pow(domain[faces].stateVar[1],2)+pow(domain[faces].stateVar[2],2))/domain[faces].stateVar[0]);
		else
		{
			presminus=(gammma[0]-1)*(domain[x].stateVar[3]-0.5*(pow(domain[x].stateVar[1],2)+pow(domain[x].stateVar[2],2))/domain[x].stateVar[0]);
		}

		double plus,minus;
		if(abs(machplus)>=1)
			plus=0.5*(1+machplus/abs(machplus));
		else
			plus=0.25*pow((machplus+1),2)*(2-machplus)+3/16*machplus*pow((pow(machplus,2)-1),2);
		if(abs(machminus)>=1)
			minus=0.5*(1-machminus/abs(machminus));
		else
			minus=0.25*pow((machminus-1),2)*(2+machminus)-3/16*machminus*pow((pow(machminus,2)-1),2);

		domain[x].presflux[y][0]=(pressplus*plus+presminus*minus)*domain[x].norms[y][0]*sqrt(pow(domain[x].nodes[(y+1)%4][0]-domain[x].nodes[y][0],2)\
		+pow(domain[x].nodes[(y+1)%4][1]-domain[x].nodes[y][1],2));
		domain[x].presflux[y][1]=(pressplus*plus+presminus*minus)*domain[x].norms[y][1]*sqrt(pow(domain[x].nodes[(y+1)%4][0]-domain[x].nodes[y][0],2)\
		+pow(domain[x].nodes[(y+1)%4][1]-domain[x].nodes[y][1],2));
	}
}
