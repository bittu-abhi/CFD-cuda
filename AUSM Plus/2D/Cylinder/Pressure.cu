#include "hip/hip_runtime.h"
#include "ausmPlus.h"
#include <cmath>
#include <algorithm>

__global__ void pressureFlux(cell *domain, double *R, double *gammma)
{
	int y=threadIdx.x;
	int x=blockIdx.x;
	int faces=(int)domain[x].face[y][0];
	int note;
	if(domain[x].flag==0 || domain[x].flag==4)
	{
		//Calculating the critical speed of sound for all the four sides/faces and the element itself
		double a_s[2];

		if(domain[x].face[y][0]<1 || domain[x].face[y][0]>26000)
			note=y;

		//Element
		a_s[0]=sqrt(2*(gammma[0]-1)/(gammma[0]+1)*(domain[x].stateVar[3]+(gammma[0]-1)*(domain[x].stateVar[3]\
			-0.5*(pow(domain[x].stateVar[1],2)+pow(domain[x].stateVar[2],2))/domain[x].stateVar[0])));
		//Side/face
		if(domain[x].flag!=4)
			a_s[1]=sqrt(2*(gammma[0]-1)/(gammma[0]+1)*(domain[faces].stateVar[3]+(gammma[0]-1)*(domain[faces].stateVar[3]\
			-0.5*(pow(domain[faces].stateVar[1],2)+pow(domain[faces].stateVar[2],2))/domain[faces].stateVar[0])));
		else
		{
			a_s[1]=a_s[0];
		}

		//speed for the boundary calculation
		a_s[0]=pow(a_s[0],2)/max(a_s[0],abs(sqrt(pow(domain[x].stateVar[1]/domain[x].stateVar[0],2)+pow(domain[x].stateVar[2]/domain[x].stateVar[0],2))));
		if(domain[x].flag!=4)
			a_s[1]=pow(a_s[0],2)/max(a_s[0],abs(sqrt(pow(domain[faces].stateVar[1]/domain[faces].stateVar[0],2)+pow(domain[faces].stateVar[2]/domain[faces].stateVar[0],2))));
		else
		{
			a_s[1]=a_s[0];
		}

		int i1,i2;
		if(domain[x].flag!=4)
		{
			for (int i = 0; i < 4; ++i)
			{
				for (int j = 0; j < 4; ++j)
				{
					if(domain[x].nodes[i][0]==domain[faces].nodes[j][0] && domain[x].nodes[i][1]==domain[faces].nodes[j][1])
					{
						if(domain[x].nodes[(i+1)%4][0]==domain[faces].nodes[(j+1)%4][0] && domain[x].nodes[(i+1)%4][1]==domain[faces].nodes[(j+1)%4][1])
						{
							i1=i;
							i2=(i+1)%4;
						}
					}
				}
			}
		}
		else
		{
			for (int i = 0; i < 4; ++i)
			{
				if(domain[x].nodes[i][2]==domain[x].face[note][1] && domain[x].nodes[(i+1)%4][2]==domain[x].face[note][2])
				{
					i1=i;
					i2=(i+1)%4;
				}
			}
		}
		//Speed of sound at facial interface
		double a_mid=min(a_s[0],a_s[1]);
		//Mach number of incoming and outgoing waves
		double machplus=(domain[x].stateVar[1]/domain[x].stateVar[0]*(domain[x].nodes[i1][0]-domain[x].nodes[i2][0])+\
		domain[x].stateVar[2]/domain[x].stateVar[0]*(domain[x].nodes[i2][1]-domain[x].nodes[i1][1]))/a_mid/sqrt(pow(domain[x].nodes[i1][0]-domain[x].nodes[i2][0],2)\
			+pow(domain[x].nodes[i2][1]-domain[x].nodes[i1][1],2));
		double machminus;
		if(domain[x].flag!=4)
		{
			machminus=-(domain[faces].stateVar[1]/domain[faces].stateVar[0]*(domain[x].nodes[i1][0]-domain[x].nodes[i2][0])+\
			domain[faces].stateVar[2]/domain[faces].stateVar[0]*(domain[x].nodes[i2][1]-domain[x].nodes[i1][1]))/a_mid/\
			sqrt(pow(domain[x].nodes[i1][0]-domain[x].nodes[i2][0],2)+pow(domain[x].nodes[i2][1]-domain[x].nodes[i1][1],2));
		}
			
		else
		{
			machminus=machplus;
		}
		
		//Pressure Fluxes
		double pressplus=(gammma[0]-1)*(domain[x].stateVar[3]-0.5*(pow(domain[x].stateVar[1],2)+pow(domain[x].stateVar[2],2))/domain[x].stateVar[0]);
		double presminus;
		if(domain[x].flag!=4)
			presminus=(gammma[0]-1)*(domain[faces].stateVar[3]-0.5*(pow(domain[faces].stateVar[1],2)+pow(domain[faces].stateVar[2],2))/domain[faces].stateVar[0]);
		else
		{
			presminus=(gammma[0]-1)*(domain[x].stateVar[3]-0.5*(pow(domain[x].stateVar[1],2)+pow(domain[x].stateVar[2],2))/domain[x].stateVar[0]);
		}
		double plus,minus;
		if(abs(machplus)>=1)
			plus=0.5*(1+machplus/abs(machplus));
		else
			plus=0.25*pow((machplus+1),2)*(2-machplus)+3/16*machplus*pow((pow(machplus,2)-1),2);
		if(abs(machminus)>=1)
			minus=0.5*(1-machminus/abs(machminus));
		else
			minus=0.25*pow((machminus-1),2)*(2+machminus)-3/16*machminus*pow((pow(machminus,2)-1),2);

		domain[x].presflux[y][0]=(pressplus*plus+presminus*minus)*(domain[x].nodes[i1][0]-domain[x].nodes[i2][0]);
		domain[x].presflux[y][1]=(pressplus*plus+presminus*minus)*(domain[x].nodes[i2][1]-domain[x].nodes[i1][1]);
	}
}
