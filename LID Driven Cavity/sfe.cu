#include "hip/hip_runtime.h"
#include "lid.h"
#include <stdio.h>


__global__ void streamfunc(double deltax, double deltay, point *pt,int points, double convergence, int *address)
{
	int i = threadIdx.x;
	int x = blockIdx.x;
	double temp = pt[i+x*points].psi;
	if(x!=0 && x!=points-1)
	{
		if(i!=0 && i!=points-1)
		{
			pt[i+x*points].psi=1/(2/(deltax*deltax)+2/(deltay*deltay))*(pt[i+x*points].w+(pt[i+1+x*points].psi+pt[i-1+x*points].psi)/(deltax*deltax)+(pt[i+(x+1)*points].psi+pt[i+(x-1)*points].psi)/(deltay*deltay));
		}
	}
	else if (x==0 || x==points-1 || i==0 || i==points-1)
	{
			pt[i+x*points].psi=0;
	}
	pt[i+x*points].error = pt[i+x*points].psi-temp;
	if((pt[i+x*points].psi-temp)<convergence)
		*address = 1;
	//if(i==points-1 && x==points-1)
	//		printf("w=%5.14lf act=%5.14lf error=%5.14lf %d %d\n",pt[i+x*points].w,pt[i+x*points].psi,pt[i+x*points].error, x, i );

}

__global__ void updateboundary(point *pt, int points, double deltay, double deltax)
{
	int x = threadIdx.x;
	int y = blockIdx.x;	
	if(x==0)	
		pt[x+y*points].w=2*(pt[x+y*points].psi-pt[x+1+y*points].psi)/(deltax*deltax);
	else if(x==points-1)
		pt[x+y*points].w=2*(pt[x+y*points].psi-pt[x-1+y*points].psi)/(deltax*deltax);
	if(y==0)
		pt[x+y*points].w=2*(pt[x+y*points].psi-pt[x+(y+1)*points].psi)/(deltay*deltay);
	else if(y==points-1)
		pt[x+y*points].w= -2/deltay+2*(pt[x+y*points].psi-pt[x+(y-1)*points].psi)/(deltay*deltay);
}	
