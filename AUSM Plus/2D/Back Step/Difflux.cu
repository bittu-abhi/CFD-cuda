#include "hip/hip_runtime.h"
#include "ausmPlus.h"
#include <stdio.h>
	
__global__ void diffusiveFlux(cell *domain,double *R, double *gammma, double *mu,double wall_temp,double *k)
{
	int x=blockIdx.x;
	int y=threadIdx.x;
	int note=-10;
	int faces=(int)domain[x].face[y]-1;
	int ourFlag=(int)domain[x].flag;
	double delu_delx=0.0,delv_delx=0.0,delu_dely=0.0,delv_dely=0.0;
	if(ourFlag==0 || ourFlag==4)
	{
		double x_cord[]={0,0},y_cord[]={0,0};
		
		if(faces<0 || faces>42860)
		{
			note=y;
		}

		int i1,i2;
		if(ourFlag==4 && y==note)
		{
			i1=note;
			i2=(note+1)%4;
			x_cord[1]=0.5*(domain[x].nodes[i1][0]+domain[x].nodes[i2][0]);
			y_cord[1]=0.5*(domain[x].nodes[i1][1]+domain[x].nodes[i2][1]);
		}

		for (int i = 0; i < 4; ++i)
		{
			if(ourFlag!=4 || (ourFlag==4 && y!=note))
			{
				//x_cordinate of the elements
				x_cord[0]+=0.25*(domain[x].nodes[i][0]);
				x_cord[1]+=0.25*(domain[faces].nodes[i][0]);
				//Y coordinate of the elements
				y_cord[0]+=0.25*(domain[x].nodes[i][1]);
				y_cord[1]+=0.25*(domain[faces].nodes[i][1]);
			}
			else
			{
				//x_cordinate of the elements
				x_cord[0]+=0.25*(domain[x].nodes[i][0]);
				//Y coordinate of the elements
				y_cord[0]+=0.25*(domain[x].nodes[i][1]);
			}
		}

		if(abs(x_cord[1]-x_cord[0])<=0.001)
		{
			delu_delx=0.0;
			delv_delx=0.0;
		}
		else
		{
			delu_delx=(domain[x].temp_var[y][1]/domain[x].temp_var[y][0]-domain[x].stateVar[1]/domain[x].stateVar[0])/(x_cord[1]-x_cord[0]);
			delv_delx=(domain[x].temp_var[y][2]/domain[x].temp_var[y][0]-domain[x].stateVar[2]/domain[x].stateVar[0])/(x_cord[1]-x_cord[0]);
		}
		if(abs(y_cord[1]-y_cord[0])<=0.001)
		{
			delu_dely=0.0;
			delv_dely=0.0;
		}
		else
		{
			delu_dely=(domain[x].temp_var[y][1]/domain[x].temp_var[y][0]-domain[x].stateVar[1]/domain[x].stateVar[0])/(y_cord[1]-y_cord[0]);
			delv_dely=(domain[x].temp_var[y][2]/domain[x].temp_var[y][0]-domain[x].stateVar[2]/domain[x].stateVar[0])/(y_cord[1]-y_cord[0]);
		}

		double tau_xx=2*mu[0]*(delu_delx-1/2*(delu_delx+delv_dely));
		double tau_yy=2*mu[0]*(delv_dely-1/2*(delu_delx+delv_dely));
		double tau_xy=mu[0]*(delu_dely+delv_delx);

		double temp[2];
		temp[0]=(gammma[0]-1)/R[0]*(domain[x].stateVar[3]-0.5*(pow(domain[x].stateVar[1],2)+pow(domain[x].stateVar[2],2))/domain[x].stateVar[0])/domain[x].stateVar[0];
		if(ourFlag!=4 || (ourFlag==4 && y!=note))
			temp[1]=(gammma[0]-1)/R[0]*(domain[x].temp_var[y][3]-0.5*(pow(domain[x].temp_var[y][1],2)\
				+pow(domain[x].temp_var[y][2],2))/domain[x].temp_var[y][0])/domain[x].temp_var[y][0];
		else
		{
			temp[1]=wall_temp;
		}

		double delT_delx,delT_dely;	
		if(abs(x_cord[1]-x_cord[0])<=0.001)
			delT_delx=0;
		else
			delT_delx=(temp[1]-temp[0])/(x_cord[1]-x_cord[0]);
		if(abs(y_cord[1]-y_cord[0])<=0.001)
			delT_dely=0;
		else
			delT_dely=(temp[1]-temp[0])/(y_cord[1]-y_cord[0]);

		double thetaX=domain[x].stateVar[1]/domain[x].stateVar[0]*tau_xx+domain[x].stateVar[2]/domain[x].stateVar[0]*tau_xy+k[0]*delT_delx;
		double thetaY=domain[x].stateVar[1]/domain[x].stateVar[0]*tau_xy+domain[x].stateVar[2]/domain[x].stateVar[0]*tau_yy+k[0]*delT_dely;

		domain[x].diffflux[y][0]=0;
		domain[x].diffflux[y][1]=(tau_xx*domain[x].norms[y][0]+tau_xy*domain[x].norms[y][1])\
		*sqrt(pow(domain[x].nodes[y][0]-domain[x].nodes[(y+1)%4][0],2)+pow(domain[x].nodes[y][1]-domain[x].nodes[(y+1)%4][1],2));
		domain[x].diffflux[y][2]=(tau_xy*domain[x].norms[y][0]+tau_yy*domain[x].norms[y][1])\
		*sqrt(pow(domain[x].nodes[y][0]-domain[x].nodes[(y+1)%4][0],2)+pow(domain[x].nodes[y][1]-domain[x].nodes[(y+1)%4][1],2));
		domain[x].diffflux[y][3]=(thetaX*domain[x].norms[y][0]+thetaY*domain[x].norms[y][1])\
		*sqrt(pow(domain[x].nodes[y][0]-domain[x].nodes[(y+1)%4][0],2)+pow(domain[x].nodes[y][1]-domain[x].nodes[(y+1)%4][1],2));

		/*if(abs(0.25*(domain[x].nodes[0][0]+domain[x].nodes[1][0]+domain[x].nodes[2][0]+domain[x].nodes[3][0])-0.375)<0.000001 && abs(0.25*(domain[x].nodes[0][1]+domain[x].nodes[1][1]+domain[x].nodes[2][1]+domain[x].nodes[3][1])-29.875)<0.00001)
			printf("upper %5.14lf %5.14lf %5.14lf %5.14lf %5.14lf %d %d %d\n",domain[x].diffflux[y][2],delu_dely,(domain[x].temp_var[y][1]/domain[x].temp_var[y][0]-domain[x].stateVar[1]/domain[x].stateVar[0])/(y_cord[1]-y_cord[0]),y_cord[1]-y_cord[0],(domain[x].temp_var[y][1]/domain[x].temp_var[y][0]-domain[x].stateVar[1]/domain[x].stateVar[0]),x,y,note );
	
		if(abs(0.25*(domain[x].nodes[0][0]+domain[x].nodes[1][0]+domain[x].nodes[2][0]+domain[x].nodes[3][0])-0.3757366911)<0.000001 && abs(0.25*(domain[x].nodes[0][1]+domain[x].nodes[1][1]+domain[x].nodes[2][1]+domain[x].nodes[3][1])-13.8545675278)<0.00001)
			printf("lower %5.14lf %5.14lf %5.14lf %5.14lf %5.14lf %d %d %d\n",domain[x].diffflux[y][2],delu_dely,(domain[x].temp_var[y][1]/domain[x].temp_var[y][0]-domain[x].stateVar[1]/domain[x].stateVar[0])/(y_cord[1]-y_cord[0]), y_cord[1]-y_cord[0],(domain[x].temp_var[y][1]/domain[x].temp_var[y][0]-domain[x].stateVar[1]/domain[x].stateVar[0]),x,y,note );*/
	}
}	