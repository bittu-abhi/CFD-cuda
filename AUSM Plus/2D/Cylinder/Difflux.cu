#include "hip/hip_runtime.h"
#include "ausmPlus.h"
#include <stdio.h>
	
__global__ void diffusiveFlux(cell *domain,float *R, float *gammma, float *mu,float wall_temp,float *k)
{
	int x=blockIdx.x;
	int y=threadIdx.x;
	int note=-10;
	int faces=(int)domain[x].face[y]-1;
	int ourFlag=(int)domain[x].flag;
	float delu_delx=0.0,delv_delx=0.0,delu_dely=0.0,delv_dely=0.0;
	if(ourFlag==0 || ourFlag==4 || ourFlag==2)
	{
		float x_cord[]={0,0},y_cord[]={0,0};
		
		if(faces<0 || faces>50266)
		{
			note=y;
		}

		int i1,i2;
		if(ourFlag==4 && y==note)
		{
			i1=note;
			i2=(note+1)%3;
			x_cord[1]=(1.0/2.0)*(domain[x].nodes[i1][0]+domain[x].nodes[i2][0]);
			y_cord[1]=(1.0/2.0)*(domain[x].nodes[i1][1]+domain[x].nodes[i2][1]);
		}

		for (int i = 0; i < 3; ++i)
		{
			if(ourFlag!=4 || (ourFlag==4 && y!=note))
			{
				//x_cordinate of the elements
				x_cord[0]+=(1.0/3.0)*(domain[x].nodes[i][0]);
				x_cord[1]+=(1.0/3.0)*(domain[faces].nodes[i][0]);
				//Y coordinate of the elements
				y_cord[0]+=(1.0/3.0)*(domain[x].nodes[i][1]);
				y_cord[1]+=(1.0/3.0)*(domain[faces].nodes[i][1]);
			}
			else
			{
				//x_cordinate of the elements
				x_cord[0]+=(1.0/3.0)*(domain[x].nodes[i][0]);
				//Y coordinate of the elements
				y_cord[0]+=(1.0/3.0)*(domain[x].nodes[i][1]);
			}
		}

		if(ourFlag==2 && y==note)
		{
			i1=note;
			i2=(note+1)%3;
			x_cord[1]=(1.0/2.0)*(domain[x].nodes[i1][0]+domain[x].nodes[i2][0]);
			y_cord[1]=(1.0/2.0)*(domain[x].nodes[i1][1]+domain[x].nodes[i2][1]);
		}
		else if(ourFlag==2 && y!=note)
		{
			for (int i = 0; i < 3; ++i)
			{
				//x_cordinate of the elements
				x_cord[0]+=(1.0/3.0)*(domain[x].nodes[i][0]);
				x_cord[1]+=(1.0/3.0)*(domain[faces].nodes[i][0]);
				//Y coordinate of the elements
				y_cord[0]+=(1.0/3.0)*(domain[x].nodes[i][1]);
				y_cord[1]+=(1.0/3.0)*(domain[faces].nodes[i][1]);
			}
		}

		if(abs(x_cord[1]-x_cord[0])<=0.001)
		{
			delu_delx=0.0;
			delv_delx=0.0;
		}
		else
		{
			delu_delx=(domain[x].temp_var[y][1]/domain[x].temp_var[y][0]-domain[x].stateVar[1]/domain[x].stateVar[0])/(x_cord[1]-x_cord[0]);
			delv_delx=(domain[x].temp_var[y][2]/domain[x].temp_var[y][0]-domain[x].stateVar[2]/domain[x].stateVar[0])/(x_cord[1]-x_cord[0]);
		}
		if(abs(y_cord[1]-y_cord[0])<=0.001)
		{
			delu_dely=0.0;
			delv_dely=0.0;
		}
		else
		{
			delu_dely=(domain[x].temp_var[y][1]/domain[x].temp_var[y][0]-domain[x].stateVar[1]/domain[x].stateVar[0])/(y_cord[1]-y_cord[0]);
			delv_dely=(domain[x].temp_var[y][2]/domain[x].temp_var[y][0]-domain[x].stateVar[2]/domain[x].stateVar[0])/(y_cord[1]-y_cord[0]);
		}

		float tau_xx=2*mu[0]*(delu_delx-1/3*(delu_delx+delv_dely));
		float tau_yy=2*mu[0]*(delv_dely-1/3*(delu_delx+delv_dely));
		float tau_xy=mu[0]*(delu_dely+delv_delx);

		float temp[2];
		temp[0]=(gammma[0]-1)/R[0]*(domain[x].stateVar[3]-0.5*(powf(domain[x].stateVar[1],2)+powf(domain[x].stateVar[2],2))/domain[x].stateVar[0])/domain[x].stateVar[0];
		if(ourFlag!=4 || (ourFlag==4 && y!=note))
			temp[1]=(gammma[0]-1)/R[0]*(domain[x].temp_var[y][3]-0.5*(powf(domain[x].temp_var[y][1],2)\
				+powf(domain[x].temp_var[y][2],2))/domain[x].temp_var[y][0])/domain[x].temp_var[y][0];
		else
		{
			temp[1]=wall_temp;
		}

		float delT_delx,delT_dely;	
		if(abs(x_cord[1]-x_cord[0])<=0.001)
			delT_delx=0;
		else
			delT_delx=(temp[1]-temp[0])/(x_cord[1]-x_cord[0]);
		if(abs(y_cord[1]-y_cord[0])<=0.001)
			delT_dely=0;
		else
			delT_dely=(temp[1]-temp[0])/(y_cord[1]-y_cord[0]);

		float thetaX=domain[x].stateVar[1]/domain[x].stateVar[0]*tau_xx+domain[x].stateVar[2]/domain[x].stateVar[0]*tau_xy+k[0]*delT_delx;
		float thetaY=domain[x].stateVar[1]/domain[x].stateVar[0]*tau_xy+domain[x].stateVar[2]/domain[x].stateVar[0]*tau_yy+k[0]*delT_dely;

		domain[x].diffflux[y][0]=0;
		domain[x].diffflux[y][1]=(tau_xx*domain[x].norms[y][0]+tau_xy*domain[x].norms[y][1])\
		*sqrt(powf(domain[x].nodes[y][0]-domain[x].nodes[(y+1)%3][0],2)+powf(domain[x].nodes[y][1]-domain[x].nodes[(y+1)%3][1],2));
		domain[x].diffflux[y][2]=(tau_xy*domain[x].norms[y][0]+tau_yy*domain[x].norms[y][1])\
		*sqrt(powf(domain[x].nodes[y][0]-domain[x].nodes[(y+1)%3][0],2)+powf(domain[x].nodes[y][1]-domain[x].nodes[(y+1)%3][1],2));
		domain[x].diffflux[y][3]=(thetaX*domain[x].norms[y][0]+thetaY*domain[x].norms[y][1])\
		*sqrt(powf(domain[x].nodes[y][0]-domain[x].nodes[(y+1)%3][0],2)+powf(domain[x].nodes[y][1]-domain[x].nodes[(y+1)%3][1],2));

		/*if(abs((1.0/3.0)*(domain[x].nodes[0][0]+domain[x].nodes[1][0]+domain[x].nodes[2][0])+0.524158619)<0.001 && abs((1.0/3.0)*(domain[x].nodes[0][1]+domain[x].nodes[1][1]+domain[x].nodes[2][1])-0.8526501336)<0.001)
		{
			printf("diffusive %5.14lf %5.14lf %5.14lf %5.14lf	 %d %d %d\n",domain[x].diffflux[y][0],domain[x].diffflux[y][1],domain[x].diffflux[y][2],domain[x].diffflux[y][3],domain[x].flag,x+1,y);
			printf("%5.14lf %5.14lf %5.14lf %5.14lf\n",x_cord[0],x_cord[1],y_cord[0],y_cord[1]);
		}*/
	}
}
