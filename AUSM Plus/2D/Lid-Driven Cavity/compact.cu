#include "hip/hip_runtime.h"
#include "lid.h"
#include <stdio.h>

__device__ void TDMA(int points, double *D, int *coef,int y_level,int flagD, int flagXY, point *pt, int flagPSIW)
{
	double mod_c[500];
	double mod_d[500];
	for (int i = 0; i < points; ++i)
	{
		if (i==0)
		{
			mod_c[i] = coef[2]/coef[1];
			mod_d[i]=D[i]/coef[1];
		}
		else if( i!=0 || i!=points-1 || i!=1 || i!=points-2)
		{
			mod_c[i] = coef[8]/(coef[7]-coef[6]*mod_c[i-1]);
			mod_d[i] = (D[i]-coef[6]*mod_d[i-1])/(coef[7]-coef[6]*mod_c[i-1]);
		}
		else if(i == 1 || i == points-2)
		{
			mod_c[i] = coef[5]/(coef[4]-coef[3]*mod_c[i-1]);
			mod_d[i] = (D[i]-coef[3]*mod_d[i-1])/(coef[4]-coef[3]*mod_c[i-1]);	
		}	
	}
	mod_d[points-1]= (D[points-1]-coef[9]*mod_d[points-2])/(coef[10]-coef[9]*mod_c[points-2]);

	for (int i = points-1; i >= 0; i=i-1)
	{
		if (i==points-1)
		{
			if(flagD==0 && flagPSIW == 0)
				pt[(flagXY==0)?(i+points*y_level):(i*points+y_level)].delW[flagXY]=mod_d[i];
			if (flagD==1 && flagPSIW == 0)
				pt[(flagXY==0)?(i+points*y_level):(i*points+y_level)].delW2[flagXY]=mod_d[i];
			if (flagPSIW == 1)
				pt[(flagXY==0)?(i+points*y_level):(i*points+y_level)].delpsi[flagXY]=mod_d[i];
			
		}
		else
		{
			if(flagD==0 && flagPSIW==0)
				pt[(flagXY==0)?(i+points*y_level):(i*points+y_level)].delW[flagXY]=mod_d[i]-mod_c[i]*pt[(flagXY==0)?(i+1+points*y_level):((i+1)*points+y_level)].delW[flagXY];
			if(flagD!=0 && flagPSIW==0)
				pt[(flagXY==0)?(i+points*y_level):(i*points+y_level)].delW2[flagXY]=mod_d[i]-mod_c[i]*pt[(flagXY==0)?(i+1+points*y_level):((i+1)*points+y_level)].delW2[flagXY];
			if (flagPSIW==1)
				pt[(flagXY==0)?(i+points*y_level):(i*points+y_level)].delpsi[flagXY]=mod_d[i]-mod_c[i]*pt[(flagXY==0)?(i+1+points*y_level):((i+1)*points+y_level)].delpsi[flagXY];
		}
	}
}

__global__ void compact1D(point *pt, int points, double delta, int flagXY, int flagPSIW)
{
	int coef[12] = {0,2,4,1,4,1,1,3,1,4,2,0};
	int y_level = blockIdx.x;
	double D[500];
	for (int i = 0; i < points; ++i)
	{
		if(flagPSIW==0)
		{
			if (i==0)
				D[i] = (-5*pt[(flagXY==0)?(i+points*y_level):(i*points+y_level)].w+4*pt[(flagXY==0)?(i+1+points*y_level):((i+1)*points+y_level)].w+pt[(flagXY==0)?(i+2+points*y_level):((i+2)*points+y_level)].w)/delta;
			else if(i==1)
				D[i] = 3/delta*(pt[(flagXY==0)?(i+1+points*y_level):((i+1)*points+y_level)].w-pt[(flagXY==0)?(i-1+points*y_level):((i-1)*points+y_level)].w);
			else if (i== points-1)
				D[i] = -1*(-5*pt[(flagXY==0)?(i+points*y_level):(i*points+y_level)].w+4*pt[(flagXY==0)?(i-1+points*y_level):((i-1)*points+y_level)].w+pt[(flagXY==0)?(i-2+points*y_level):((i-2)*points+y_level)].w)/delta;
			else if (i==points-2)
				D[i] = -3/delta*(pt[(flagXY==0)?(i-1+points*y_level):((i-1)*points+y_level)].w-pt[(flagXY==0)?(i+1+points*y_level):((i+1)*points+y_level)].w);
			else
				D[i] = (-pt[(flagXY==0)?(i-2+points*y_level):((i-2)*points+y_level)].w-28*pt[(flagXY==0)?(i-1+points*y_level):((i-1)*points+y_level)].w+28*pt[(flagXY==0)?(i+1+points*y_level):((i+1)*points+y_level)].w+pt[(flagXY==0)?(i+2+points*y_level):((i+2)*points+y_level)].w)/(12*delta);
		}
		else
		{
			if (i==0)
				D[i] = (-5*pt[(flagXY==0)?(i+points*y_level):(i*points+y_level)].psi+4*pt[(flagXY==0)?(i+1+points*y_level):((i+1)*points+y_level)].psi+pt[(flagXY==0)?(i+2+points*y_level):((i+2)*points+y_level)].psi)/delta;
			else if(i==1)
				D[i] = 3/delta*(pt[(flagXY==0)?(i+1+points*y_level):((i+1)*points+y_level)].psi-pt[(flagXY==0)?(i-1+points*y_level):((i-1)*points+y_level)].psi);
			else if (i== points-1)
				D[i] = -1*(-5*pt[(flagXY==0)?(i+points*y_level):(i*points+y_level)].psi+4*pt[(flagXY==0)?(i-1+points*y_level):((i-1)*points+y_level)].psi+pt[(flagXY==0)?(i-2+points*y_level):((i-2)*points+y_level)].psi)/delta;
			else if (i==points-2)
				D[i] = -3/delta*(pt[(flagXY==0)?(i-1+points*y_level):((i-1)*points+y_level)].psi-pt[(flagXY==0)?(i+1+points*y_level):((i+1)*points+y_level)].psi);
			else
				D[i] = (-pt[(flagXY==0)?(i-2+points*y_level):((i-2)*points+y_level)].psi-28*pt[(flagXY==0)?(i-1+points*y_level):((i-1)*points+y_level)].psi+28*pt[(flagXY==0)?(i+1+points*y_level):((i+1)*points+y_level)].psi+pt[(flagXY==0)?(i+2+points*y_level):((i+2)*points+y_level)].psi)/(12*delta);
		}
		/*if(y_level==points-1 && i==points-1)
		{
			printf("%lf delpsix=%lf delpsiy=%lf delwx=%lf delwy=%lf delw2x=%5.14lf delw2y=%5.14lf \n",D[i],pt[i+y_level*points].delpsi[0],pt[i+y_level*points].delpsi[1],pt[i+y_level*points].delW[0],pt[i+y_level*points].delW[1],pt[i+y_level*points].delW2[0],pt[i+y_level*points].delW2[1]);
		}*/		
		
	}
	TDMA(points,D,coef,y_level,0,flagXY,pt,flagPSIW);
}

__global__ void compact2D(point *pt, int points, double delta, int flagXY)
{
	int coef[12] = {0,1,11,1,10,1,2/11,1,2/11,11,1,0};
	int y_level = blockIdx.x;
	double D[500];
	for (int i = 0; i < points; ++i)
	{
		if (i==0)
			D[i] = (13*pt[(flagXY==0)?(i+points*y_level):(i*points+y_level)].w-27*pt[(flagXY==0)?(i+1+points*y_level):((i+1)*points+y_level)].w+15*pt[(flagXY==0)?(i+2+points*y_level):((i+2)*points+y_level)].w-pt[(flagXY==0)?(i+3+points*y_level):((i+3)*points+y_level)].w)/(delta*delta);
		else if(i==1 || i==points-2)
			D[i] = 12/(delta*delta)*(pt[(flagXY==0)?(i+1+points*y_level):((i+1)*points+y_level)].w-2*pt[(flagXY==0)?(i+points*y_level):(i*points+y_level)].w+pt[(flagXY==0)?(i-1+points*y_level):((i-1)*points+y_level)].w);
		else if (i== points-1)
			D[i] = 1 /(delta*delta)*(13*pt[(flagXY==0)?(i+points*y_level):(i*points+y_level)].w-27*pt[(flagXY==0)?(i-1+points*y_level):((i-1)*points+y_level)].w+15*pt[(flagXY==0)?(i-2+points*y_level):((i-2)*points+y_level)].w-pt[(flagXY==0)?(i-3+points*y_level):((i-3)*points+y_level)].w);
		else
			D[i] = 3/(11*4*delta*delta)*(pt[(flagXY==0)?(i-2+points*y_level):((i-2)*points+y_level)].w-2*pt[(flagXY==0)?(i+points*y_level):(i*points+y_level)].w+pt[(flagXY==0)?(i+2+points*y_level):((i+2)*points+y_level)].w)+12/(11*delta*delta)*(pt[(flagXY==0)?(i-1+points*y_level):((i-1)*points+y_level)].w-2*pt[(flagXY==0)?(i+points*y_level):(i*points+y_level)].w+pt[(flagXY==0)?(i+1+points*y_level):((i+1)*points+y_level)].w);
	}
	TDMA(points,D,coef,y_level,1,flagXY,pt,0);
}