#include "hip/hip_runtime.h"
#include "ausmPlus.h"
#include <hip/hip_runtime_api.h>
#include <math.h>
#include <stdio.h>

__global__ void set_nodes(double *node, cell *domain, double *boundary, double *initial)
{
	int x=blockIdx.x;
	int y=threadIdx.x;
	int flag1=0;
	int temp=3*((int)(domain[x].nodes[y][2])-1);
	domain[x].nodes[y][1]=node[temp+1];
	domain[x].nodes[y][0]=node[temp];
	
	for(int i=0;i<1040*2;i++)
	{
		if(domain[x].nodes[y][2]==boundary[i])
		{
			flag1=1;
			break;
		}
	}

	if(flag1==0)
		domain[x].flag=0;

	if(domain[x].nodes[y][2]>521 && domain[x].nodes[y][2]<1041)
	{
		domain[x].flag=2;
	}
	
	if((domain[x].nodes[y][2]>66 && domain[x].nodes[y][2]<522) || (domain[x].nodes[y][2]<1041 && domain[x].nodes[y][2]>641) || domain[x].nodes[y][2]==522)
			domain[x].flag=4;

	if(domain[x].nodes[y][2]>=1 && domain[x].nodes[y][2]<67)
	{
		domain[x].flag=1;
		domain[x].stateVar[0]=initial[0];
		domain[x].stateVar[1]=initial[1];
		domain[x].stateVar[2]=initial[2];
		domain[x].stateVar[3]=initial[3];	
	}
}

__global__ void set_neighbour(cell *domain)
{
	int x=blockIdx.x;
	int y=threadIdx.x;
	int i,flag1=0,flag2=0;
	for (i = 0; i < 42830; i++)
	{
		for (int j = 0; j < 4; j+=1)
		{
			if(abs(domain[i].nodes[j][0]-domain[x].nodes[y][0])<0.0001 && i!=x && abs(domain[i].nodes[j][1]-domain[x].nodes[y][1])<0.0001)
				flag1=1;
			if(abs(domain[i].nodes[j][0]-domain[x].nodes[(y+1)%4][0])<0.0001 && i!=x && abs(domain[i].nodes[j][1]-domain[x].nodes[(y+1)%4][1])<0.0001)
				flag2=1;
		}
		if(flag1==1 && flag2==1)
		{	
			domain[x].face[y]=i+1;
			break;
		}
		flag1=0;
		flag2=0;
	}
}

__global__ void calculate_norm(cell *domain)
{
	int x=blockIdx.x;
	int y=threadIdx.x;

	//Now to determine if the normal is pointing outward, and if not, then change accordingly
	double cen_cord[2];
	cen_cord[0]=0.250000*(domain[x].nodes[0][0]+domain[x].nodes[1][0]+domain[x].nodes[2][0]+domain[x].nodes[3][0]);
	cen_cord[1]=0.250000*(domain[x].nodes[0][1]+domain[x].nodes[1][1]+domain[x].nodes[2][1]+domain[x].nodes[3][1]);

	//construct the face
	double m,c;
	m=(domain[x].nodes[(y+1)%4][1]-domain[x].nodes[y][1])/(domain[x].nodes[(y+1)%4][0]-domain[x].nodes[y][0]);

	c=domain[x].nodes[y][1]-m*domain[x].nodes[y][0];

	//A perpendicular line passing through the centre of the element
	if(m!=0.0000 && !isinf(m))
	{
		double req_m=-1/m;
		double req_c=cen_cord[1]-req_m*cen_cord[0];

		//Intersection of this line with the face would give a point on the face. Now using this point as (x1,y2), we would
		//always get a vector pointing outward from the face,regardless of the way the nodes are number(clockwise or anticlockwise)
		double req_x=(c-req_c)/(req_m-m);
		double req_y=m*req_x+c;
		
		domain[x].norms[y][0]=(req_x-cen_cord[0]);
		domain[x].norms[y][1]=(req_y-cen_cord[1]);

		double dino=sqrt(pow((req_x-cen_cord[0]),2)+pow((req_y-cen_cord[1]),2));
		
		domain[x].norms[y][0]/=dino;
		domain[x].norms[y][1]/=dino;		

	}
	else if(m==0.0000)
	{
		domain[x].norms[y][0]=0;
		if(domain[x].nodes[y][1]<cen_cord[1])
			domain[x].norms[y][1]=-1.000;
		else 
			domain[x].norms[y][1]=1.000;
	}
	else
	{
		domain[x].norms[y][1]=0;
		if(domain[x].nodes[y][0]<cen_cord[0])
			domain[x].norms[y][0]=-1.000;
		else
			domain[x].norms[y][0]=1.0000;
	}
}

__global__ void read_values(cell *domain)
{
	int x=blockIdx.x;
	int y=threadIdx.x;
	int faces=(int)domain[x].face[y]-1;
	int note=-10;

	if(faces<0 || faces >42829)
	{
		note=y;
	}
	if(y!=note)
	{
		for (int i = 0; i < 4; ++i)
		{
			domain[x].temp_var[y][i]=domain[faces].stateVar[i];
		}
	}
	else
	{
		if(domain[x].flag==4)
		{
			domain[x].temp_var[note][0]=1.225;
			domain[x].temp_var[note][1]=-1.0000*domain[x].stateVar[1];
			domain[x].temp_var[note][2]=-1.0000*domain[x].stateVar[2];
			domain[x].temp_var[note][3]=domain[x].stateVar[3];
			//printf("%lf %lf %lf %lf %d %d %d\n",domain[x].stateVar[1],domain[x].temp_var[y][1],domain[x].stateVar[2],domain[x].temp_var[y][2],note, x,y);
		}
	}
}