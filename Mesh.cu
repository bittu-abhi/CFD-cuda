#include "hip/hip_runtime.h"
#include "ausmPlus.h"

__global__ void set_nodes(double *node, cell *domain, double *boundary)
{
	int x=blockIdx.x;
	int y=threadIdx.x;
	int flag1=0;
	int temp=3*((int)(domain[x].nodes[y][2])-1);
	domain[x].nodes[y][1]=node[temp+1];
	domain[x].nodes[y][0]=node[temp];
	if(domain[x].nodes[y][2]>126 && domain[x].nodes[y][2]<176)
	{
		domain[x].flag=1;	
	}
	else if(domain[x].nodes[y][2]>=25402 && domain[x].nodes[y][2]<=25452)
	{
		domain[x].flag=2;
	}
	for(int i=0;i<599*2;i++)
	{
		if(domain[x].nodes[0][2]==boundary[i] || domain[x].nodes[1][2]==boundary[i] || domain[x].nodes[2][2]==boundary[i] || domain[x].nodes[3][2]==boundary[i])
		{
			flag1=1;
			break;
		}
	}
	if(flag1==0)
		domain[x].flag=0;

	for (int i = 0; i < 4; ++i)
	{
		if(abs(domain[x].nodes[i][0]-(20+sqrt(4-pow(domain[x].nodes[i][1]-15,2))))<0.002 || abs(domain[x].nodes[i][0]-(20-sqrt(4-pow(domain[x].nodes[i][1]-15,2))))<0.002)
			domain[x].flag=4;
	}
	if(flag1==1 && domain[x].flag!=1 && domain[x].flag!=2 && domain[x].flag!=4 && domain[x].flag!=0)
			domain[x].flag=3;
}

__global__ void set_neighbour(cell *domain)
{
	int x=blockIdx.x;
	int y=threadIdx.x;
	int i,flag1=0,flag2=0;
	for (i = 0; i < 25000; i+=1)
	{
		for (int j = 0; j < 4; j+=1)
		{
			if(domain[i].nodes[j][2]==domain[x].nodes[y][2] && i!=x)
				flag1=1;
			if(domain[i].nodes[j][2]==domain[x].nodes[(y+1)%4][2] && i!=x)
				flag2=1;
		}
		if(flag1==1 && flag2==1)
		{	
			domain[x].face[y][0]=i;
			domain[x].face[y][1]=domain[x].nodes[y][2];
			domain[x].face[y][2]=domain[x].nodes[(y+1)%4][2];
			break;
		}
		flag1=0;
		flag2=0;
	}
}