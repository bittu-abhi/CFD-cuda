#include "sbli.h"

point::point()
{}

void sbli(point *pt, int points, double timesteps, double delta)
{
	int x = blockIdx.x;
	int y = blockIdx.y;
	int state = threadIdx.x;

	//CPU variables
	point *ptr = new point[points*points];
	int flag=0;

	//GPU variables
	point *d_pt;
	int *d_flag;

	hipMalloc((void **)&d_pt,points*points*sizeof(double));
	hipMalloc((void **)&d_flag,sizeof(int));

	hipStream_t stream1;
	hipStreamCreate(&stream1);

	hipMemcpy(d_pt,&ptr[0],points*points*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(d_flag,&flag[0],sizeof(int),hipMemcpyHostToDevice);

	WENO<<<points,points,4>>>(pt,points,0);
	WENO<<<points,points,4>>>(pt,points,1);
	pt[x+y*points]
}
