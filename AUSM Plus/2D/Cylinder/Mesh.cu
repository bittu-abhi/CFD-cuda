#include "hip/hip_runtime.h"
#include "ausmPlus.h"
#include <hip/hip_runtime_api.h>
#include <math.h>
#include <stdio.h>

__global__ void set_nodes(double *node, cell *domain, double *boundary,double *initial)
{
	int x=blockIdx.x;
	int y=threadIdx.x;
	int flag1=0;
	int temp=3*((int)(domain[x].nodes[y][2])-1);
	domain[x].nodes[y][1]=node[temp+1];
	domain[x].nodes[y][0]=node[temp];
	
	if(domain[x].nodes[y][2]>126 && domain[x].nodes[y][2]<176)
	{
		domain[x].flag=1;
		domain[x].stateVar[0]=initial[0];
		domain[x].stateVar[1]=initial[1];
		domain[x].stateVar[2]=initial[2];
		domain[x].stateVar[3]=initial[3];	
	}
	
	for(int i=0;i<599*2;i++)
	{
		if(domain[x].nodes[0][2]==boundary[i] || domain[x].nodes[1][2]==boundary[i] || domain[x].nodes[2][2]==boundary[i] || domain[x].nodes[3][2]==boundary[i])
		{
			flag1=1;
			break;
		}
	}
	if(flag1==0)
		domain[x].flag=0;

	for (int i = 0; i < 4; ++i)
	{
		if(abs(domain[x].nodes[i][0]-(20+sqrt(4-pow(domain[x].nodes[i][1]-15,2))))<0.005 || abs(domain[x].nodes[i][0]-(20-sqrt(4-pow(domain[x].nodes[i][1]-15,2))))<0.005)
			domain[x].flag=4;
	}
	if(flag1==1 && domain[x].flag!=1 && domain[x].flag!=4 && domain[x].flag!=0)
	{
		domain[x].flag=3;
		domain[x].stateVar[0]=initial[0];
		domain[x].stateVar[1]=initial[1];
		domain[x].stateVar[2]=initial[2];
		domain[x].stateVar[3]=initial[3];
	}

	if(domain[x].nodes[y][2]>=25402 && domain[x].nodes[y][2]<=25452)
	{
		domain[x].flag=2;
	}
}

__global__ void set_neighbour(cell *domain)
{
	int x=blockIdx.x;
	int y=threadIdx.x;
	int i,flag1=0,flag2=0;
	for (i = 0; i < 25000; i+=1)
	{
		for (int j = 0; j < 4; j+=1)
		{
			if(domain[i].nodes[j][0]==domain[x].nodes[y][0] && i!=x && domain[i].nodes[j][1]==domain[x].nodes[y][1])
				flag1=1;
			if( domain[i].nodes[j][0]==domain[x].nodes[(y+1)%4][0] && i!=x && domain[i].nodes[j][1]==domain[x].nodes[(y+1)%4][1])
				flag2=1;
		}
		if(flag1==1 && flag2==1)
		{	
			domain[x].face[y]=i+1;
			break;
		}
		flag1=0;
		flag2=0;
	}
}

__global__ void calculate_norm(cell *domain)
{
	int x=blockIdx.x;
	int y=threadIdx.x;

	//Now to determine if the normal is pointing outward, and if not, then change accordingly
	double cen_cord[2];
	cen_cord[0]=0.250000*(domain[x].nodes[0][0]+domain[x].nodes[1][0]+domain[x].nodes[2][0]+domain[x].nodes[3][0]);
	cen_cord[1]=0.250000*(domain[x].nodes[0][1]+domain[x].nodes[1][1]+domain[x].nodes[2][1]+domain[x].nodes[3][1]);

	//construct the face
	double m,c;
	m=(domain[x].nodes[(y+1)%4][1]-domain[x].nodes[y][1])/(domain[x].nodes[(y+1)%4][0]-domain[x].nodes[y][0]);

	c=domain[x].nodes[y][1]-m*domain[x].nodes[y][0];

	//A perpendicular line passing through the centre of the element
	if(m!=0.0000 && !isinf(m))
	{
		double req_m=-1/m;
		double req_c=cen_cord[1]-req_m*cen_cord[0];

		//Intersection of this line with the face would give a point on the face. Now using this point as (x1,y2), we would
		//always get a vector pointing outward from the face,regardless of the way the nodes are number(clockwise or anticlockwise)
		double req_x=(c-req_c)/(req_m-m);
		double req_y=m*req_x+c;
		
		domain[x].norms[y][0]=(req_x-cen_cord[0]);
		domain[x].norms[y][1]=(req_y-cen_cord[1]);

		double dino=sqrt(pow((req_x-cen_cord[0]),2)+pow((req_y-cen_cord[1]),2));
		
		domain[x].norms[y][0]/=dino;
		domain[x].norms[y][1]/=dino;		

	}
	else if(m==0.0000)
	{
		domain[x].norms[y][0]=0;
		if(domain[x].nodes[y][1]<cen_cord[1])
			domain[x].norms[y][1]=-1.000;
		else 
			domain[x].norms[y][1]=1.000;
	}
	else
	{
		domain[x].norms[y][1]=0;
		if(domain[x].nodes[y][0]<cen_cord[0])
			domain[x].norms[y][0]=-1.000;
		else
			domain[x].norms[y][0]=1.0000;
	}
	double te=domain[x].norms[y][1];
	if(abs(domain[x].norms[y][1]-1)<0.0001)
	{
		domain[x].norms[y][1]=te/abs(te);
		domain[x].norms[y][0]=0;
	}
	te=domain[x].norms[y][0];
	if(abs(domain[x].norms[y][0]-1)<0.0001)
	{
		domain[x].norms[y][0]=te/abs(te);
		domain[x].norms[y][1]=0;
	}
}

__global__ void read_values(cell *domain)
{
	int x=blockIdx.x;
	int y=threadIdx.x;
	int faces=(int)domain[x].face[y]-1;
	int note=-10;

	if(faces<0 || faces >25000)
	{
		note=y;
	}
	if(y!=note)
	{
		for (int i = 0; i < 4; ++i)
		{
			domain[x].temp_var[y][i]=domain[faces].stateVar[i];
		}
	}
	else
	{
		if(domain[x].flag==4)
		{
			domain[x].temp_var[note][0]=1.225;
			domain[x].temp_var[note][1]=-1.0000*domain[x].stateVar[1];
			domain[x].temp_var[note][2]=-1.0000*domain[x].stateVar[2];
			domain[x].temp_var[note][3]=domain[x].stateVar[3];
			//printf("%lf %lf %lf %lf %d %d %d\n",domain[x].stateVar[1],domain[x].temp_var[y][1],domain[x].stateVar[2],domain[x].temp_var[y][2],note, x,y);
		}
	}
}
