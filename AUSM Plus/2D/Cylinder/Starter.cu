#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cmath>
#include "ausmPlus.h"

using namespace std;

double gammma;
double mu;
double k;
double R;	

int main()
{
	gammma=1.4;
	mu=pow(1.798,-5);
	k=0.0251;
	R=286.9;
	
	double initial[4];
	//Rho
	initial[0]=1.225;
	//Rho*U
	initial[1]=2;
	//Rho *V
	initial[2]=0;
	//Rho*E, E is the internal energy including the kinetic energy(i.e. total intenal energy)
	initial[3]=101325.;
	//Time steps and delta_t
	double timesteps=1;
	double deltat=0.0001;
	ausmplus(initial,timesteps,deltat);

	return 0;
}