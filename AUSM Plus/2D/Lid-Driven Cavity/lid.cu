#include "hip/hip_runtime.h"
#include "lid.h"
#include <iostream>

using namespace std;

__global__ void initial(point *pt , int points)
{
	int x = blockIdx.x;
	int y = threadIdx.x;
	pt[y+x*sizeof(point)].psi = 0;
	pt[y+x*sizeof(point)].w = 0;
	for (int i = 0; i < 2; ++i)
	{
		pt[y+x*sizeof(point)].delW[i]=0;
		pt[y+x*sizeof(point)].delW2[i]=0;
		pt[y+x*sizeof(point)].delpsi[i]=0;
	}
}

void LID(double deltax, double deltay, double deltat,int points, double convergence, int iterations, double reynold, double timesteps)
{
	//CPU variables
	point *ptr = new point[points*points];
	int flag=0;

	//Gpu variables
	point *d_ptr;
	int *d_flag;
	hipStream_t stream1, stream2,stream3,stream4,stream5,stream6;
	hipStreamCreate(&stream1);
	hipStreamCreate(&stream2);
	hipStreamCreate(&stream3);
	hipStreamCreate(&stream4);
	hipStreamCreate(&stream5);
	hipStreamCreate(&stream6);

	//Memory allocation on GPU
	hipMalloc((void **)&d_ptr,points*points*sizeof(point));
	hipMalloc((void **)&d_flag,sizeof(int));
	//Copying memory from CPU to GPU
	hipMemcpy(d_ptr,&ptr[0],points*points*sizeof(point),hipMemcpyHostToDevice);
	hipMemcpy(d_flag,&flag,sizeof(int),hipMemcpyHostToDevice);
	cout<<"Memory copy on the GPU : done"<<endl;
	cout<<endl;

	initial<<<points,points>>>(d_ptr,points);
	hipDeviceSynchronize();

	cout<<"initialisation done..."<<endl;
	cout<<endl;

	int iter = 0;

	for (double t = 0; t < timesteps*deltat; t=t+deltat)
	{
		if((int)(t/deltat)%10000==0)
		{
			hipMemcpyAsync(&ptr[0],d_ptr,points*points*sizeof(point),hipMemcpyDeviceToHost,stream4);
			visu(ptr,t,deltat,points,deltax,deltay);
		}
		updateboundary<<<points,points,0,stream1>>>(d_ptr,points,deltay,deltax);
		cout<<"vorticity boundary updated"<<endl;
		//Streamfunction evalation
		//Flag set for convergence (1 means error is greater than required threshold)

		//Calculates the total number of iteratios
		iter = 0;
		flag = 0;
		hipMemcpy(d_flag,&flag,sizeof(int),hipMemcpyHostToDevice);

		while(flag==0 && iter<iterations)
		{
			streamfunc<<<points,points>>>(deltax,deltay,d_ptr,points,convergence,d_flag);
			hipDeviceSynchronize();
			iter=iter+1;
			hipMemcpyAsync(&flag,d_flag,sizeof(int),hipMemcpyDeviceToHost);
		}
		cout<<endl;
		/*hipDeviceSynchronize();
		cout<<"hehe"<<endl;
		secondW<<<points,points,1,stream1>>>(d_ptr,points,deltax,0);
		secondW<<<points,points,1,stream2>>>(d_ptr,points,deltay,1);*/
		//1st derivative of w
		compact1D<<<points,1,0,stream1>>>(d_ptr,points,deltax,0,0);
		compact1D<<<points,1,0,stream2>>>(d_ptr,points,deltay,1,0);
		//2nd derivative of w
		compact2D<<<points,1,0,stream3>>>(d_ptr,points,deltax,0);
		compact2D<<<points,1,0,stream4>>>(d_ptr,points,deltay,1);
		//1st derivative of psi
		compact1D<<<points,1,0,stream5>>>(d_ptr,points,deltax,0,1);
		compact1D<<<points,1,0,stream6>>>(d_ptr,points,deltay,1,1);
		hipDeviceSynchronize();
		timeW<<<points,points>>>(d_ptr,deltat,reynold,points);
		hipDeviceSynchronize();
		cout<<"t="<<t<<endl;
	}


	cout<<endl;
	hipMemcpyAsync(&ptr[0],d_ptr,points*points*sizeof(point),hipMemcpyDeviceToHost);

	cout<<"Copying final values on the CPU from GPU : done"<<endl;
	cout<<endl;
	visu(ptr,timesteps*deltat,deltat,points,deltax,deltay);
	hipStreamDestroy(stream1);
	hipStreamDestroy(stream2);
	hipStreamDestroy(stream3);	
	hipStreamDestroy(stream4);	
	hipStreamDestroy(stream5);	
	hipStreamDestroy(stream6);	

	hipFree(d_ptr);
	delete[] ptr;
}